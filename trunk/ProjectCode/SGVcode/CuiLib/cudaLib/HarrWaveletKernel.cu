#include "hip/hip_runtime.h"

#include "hip/device_functions.h"
//#include<stdio.h>
#include<assert.h>
/*------------------------------------------------------------------------------------------*/
/**
*
*
*
*/
/*------------------------------------------------------------------------------------------*/
extern "C" bool cudaInit_CUI(void);
/*------------------------------------------------------------------------------------------*/
/**
*
*
*
*/
/*------------------------------------------------------------------------------------------*/
extern "C" bool cudaDeInit_CUI(void);
/*------------------------------------------------------------------------------------------*/
/**
*
*
*
*/
/*------------------------------------------------------------------------------------------*/
extern "C" void cudaGetLastError_Sync_CUI(void);
/*------------------------------------------------------------------------------------------*/
/**
*
*
*
*/
/*------------------------------------------------------------------------------------------*/
__global__ void HarrTransformLet_MatrixSub_thread(
	unsigned char *matrixA, 
	unsigned char *matrixB, 
	int height, 
	int width, 
	unsigned char *result)
{
	int x =blockIdx.x*blockDim.x+threadIdx.x;
	int y =blockIdx.y*blockDim.y+threadIdx.y;
	if(x>=0&&x<width&&y>=0&&y<height){
		int index=y * width + x;			
		result[index]= matrixA[index] - matrixB[index] + 128; 
	}

}
/*------------------------------------------------------------------------------------------*/
/**
*
*
*
*/
/*------------------------------------------------------------------------------------------*/
void HarrTransformLet_MatrixSub_gpu(
	unsigned char *matrixA, 
	unsigned char *matrixB, 
	int height, 
	int width, 
	unsigned char *result)
{
	unsigned char *matixaA_dev;
	unsigned char *matrixB_dev;
	unsigned char *result_dev;
	hipError_t cudaStatus;
	assert(cudaInit_CUI()==true);
	//////////////////////////////////////////////////////////////////////////////
	cudaStatus=hipMalloc((void**)&matixaA_dev, width*height*sizeof(unsigned char));
	cudaStatus=hipMalloc((void**)&matrixB_dev,width*height*sizeof(unsigned char));
	cudaStatus=hipMalloc((void**)&result_dev,width*height*sizeof(unsigned char));

	cudaStatus = hipMemcpy(matixaA_dev,matrixA, width*height*sizeof(unsigned char), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(matrixB_dev,matrixB,width*height*sizeof(unsigned char), hipMemcpyHostToDevice);
	dim3    threadsPerBlock(16,16);
	dim3    numBlock((width+threadsPerBlock.x-1)/threadsPerBlock.x,(height+threadsPerBlock.y-1)/threadsPerBlock.y);
	HarrTransformLet_MatrixSub_thread<<<numBlock,threadsPerBlock>>>(
											matixaA_dev,
											matrixB_dev,											
											height,
											width,
											result_dev);
	cudaGetLastError_Sync_CUI();
	hipMemcpy(result,result_dev,width*height*sizeof(unsigned char),hipMemcpyDeviceToHost);

	hipFree(matixaA_dev);
	hipFree(matrixB_dev);
	hipFree(result_dev);
	////////////////////////////////////////////////////////////////////////////
	assert(cudaDeInit_CUI()==true);

}
/*------------------------------------------------------------------------------------------*/
/**
*
*
*
*/
/*------------------------------------------------------------------------------------------*/

/*------------------------------------------------------------------------------------------*/
/**
*
*
*
*/
/*------------------------------------------------------------------------------------------*/