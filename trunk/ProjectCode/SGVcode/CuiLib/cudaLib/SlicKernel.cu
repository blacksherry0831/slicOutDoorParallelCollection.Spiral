#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"
#include <stdio.h>
//#include<stdio.h>
#include<assert.h>
#ifndef UINT32
typedef unsigned int UINT32;
#endif
#if linux||__linux||__linux__ || __GNUC__

#ifndef nullptr
#define nullptr 0
#endif

#ifndef _DEBUG
#define  _DEBUG 1
#endif
#include <unistd.h>    
#include <sys/types.h>  
#include <sys/syscall.h>//Linux system call for thread id
#include <pthread.h>

#endif



#ifdef _MSC_VER

#include <windows.h>

#endif

/*------------------------------------------------------------------------------------------*/
/**
*ʱ��2014- 11-4
*
*
*/
/*------------------------------------------------------------------------------------------*/
extern "C" void PerformSuperpixelSLIC_gpu(
	float alpha,
	float betta,
	float gama,
	float fai,
	const int kseedsSize,
	const int m_width,
	const int m_height,
	double*				m_lvec,
	double*				m_avec,
	double*				m_bvec,
	double*				host_kseedsl,
	double*				host_kseedsa,
	double*				host_kseedsb,
	double*				host_kseedsx,
	double*				host_kseedsy,
	int*				host_klabels,
	const int			STEP,
	double*				host_edgemag,
	const double		M);
/*------------------------------------------------------------------------------------------*/
extern "C" void PerformSuperpixelSLIC_ThetaMLXY_gpu_simplify(
	float alpha,
	float betta,
	float gama,
	float fai,
	const int kseedsSize,
	const int m_width,   
	const int m_height,
	double*     host_sita_n,
	double*     host_m_n,
	double*     host_L_n,
	double*     host_X_n,
	double*     host_Y_n,
	double*     host_kseedsTheta,
	double*     host_kseedsM,
	double*     host_kseedsL,
	double*     host_kseedsX,
	double*     host_kseedsY,
	int*		host_klabels,
	const int			STEP);
/*------------------------------------------------------------------------------------------*/
/**
*ʱ��2014- 11-4
*
*
*/
/*------------------------------------------------------------------------------------------*/
extern "C" void PerformSuperpixelSLIC_gpu_simplify(
	float alpha,
	float betta,
	float gama,
	float fai,
	const int kseedsSize,
	const int m_width,
	const int m_height,
	double*				m_lvec,
	double*				m_avec,
	double*				m_bvec,
	double*				host_kseedsl,
	double*				host_kseedsa,
	double*				host_kseedsb,
	double*				host_kseedsx,
	double*				host_kseedsy,
	int*				host_klabels,
	const int			STEP,
	double*				host_edgemag,
	const double		M);
extern "C" void PerformSuperpixelSLIC_gpu_simplify2(
	float alpha,
	float betta,
	float gama,
	float fai,
	const int kseedsSize,
	const int m_width,
	const int m_height,
	double*				m_lvec,
	double*				m_avec,
	double*				m_bvec,
	double*				host_kseedsl,
	double*				host_kseedsa,
	double*				host_kseedsb,
	double*				host_kseedsx,
	double*				host_kseedsy,
	int*				host_klabels,
	const int			STEP,
	double*				host_edgemag,
	const double		M);
/*------------------------------------------------------------------------------------------*/
/**
*
*
*
*/
/*------------------------------------------------------------------------------------------*/
extern "C" void DrawContoursAroundSegments_gpu(
	unsigned int*			ubuff,
	int*					labels,
	const int				width,
	const int				height,
	const unsigned int	color_in,
	const unsigned int  color_out);
/*------------------------------------------------------------------------------------------*/
/**
*
*
*
*/
/*------------------------------------------------------------------------------------------*/
extern "C" void  GetSeedsLabxy_gpu(
	const int kseedsSize,
	const int m_width,
	const int m_height,
	double*				host_m_lvec,
	double*				host_m_avec,
	double*				host_m_bvec,
	double*				host_kseedsl,
	double*				host_kseedsa,
	double*				host_kseedsb,
	double*				host_kseedsx,
	double*				host_kseedsy,
	int*				host_klabels);
/*------------------------------------------------------------------------------------------*/
/**
*
*
*
*/
/*------------------------------------------------------------------------------------------*/
extern "C" int GetThreadIdSelfwinlinux(void);
/*------------------------------------------------------------------------------------------*/
/**
*
*
*
*/
/*------------------------------------------------------------------------------------------*/
extern "C" void Get_Nighbour_E_matrix_gpu(
	int*			labels,
	const  int		NumLabels,
	UINT32 *		Matrix_E,
	const int		width,
	const int		height);
/*------------------------------------------------------------------------------------------*/
/**
*
*
*
*/
/*------------------------------------------------------------------------------------------*/
void InitMaskTemplate(
	unsigned int*			dev_mask,
	int*					dev_labels,
	const int				width,
	const int				height,
	const int                Pixel);
void StatisticsThetaMLXY_Seed_collect_CPU_ALL(
	const int			kseedsSize,
	const int			width,
	const int			height,
	double*       host_sita_n,
	double*       host_m_n,
	double*       host_L_n,
	double*       host_X_n,
	double*       host_Y_n,	
	int*				host_klabels,
	unsigned int*       host_img_mask,
    double*	      host_kseed_sita_n,
	double*	      host_kseed_m_n,
	double*	      host_kseed_L_n,
	double*       host_kseed_X_n,
	double*	      host_kseed_Y_n);
/*------------------------------------------------------------------------------------------*/
/**
*
*
*
*/
/*------------------------------------------------------------------------------------------*/
extern "C" bool cudaInit_CUI(void);
/*------------------------------------------------------------------------------------------*/
/**
*
*
*
*/
/*------------------------------------------------------------------------------------------*/
extern "C" bool cudaDeInit_CUI(void);
/*------------------------------------------------------------------------------------------*/
/**
*
*
*
*/
/*------------------------------------------------------------------------------------------*/
extern "C" void cudaGetLastError_Sync_CUI(void);
/*------------------------------------------------------------------------------------------*/
/**
*
*
*
*/
/*------------------------------------------------------------------------------------------*/
#define  NULL_MATH    1
#define  MATH_USE  0
/*------------------------------------------------------------------------------------------*/
/**
*
*
*
*/
/*------------------------------------------------------------------------------------------*/
__device__ int a;
/*------------------------------------------------------------------------------------------*/
/**
*
*
*
*/
/*------------------------------------------------------------------------------------------*/
//__inline__ __device__ double sqrt(double x)
//{
//#if 0
//	return 0;
//#else
//	/*return sqrt(x);*/
//#endif
//}
/*------------------------------------------------------------------------------------------*/
/**
*
*
*
*/
/*------------------------------------------------------------------------------------------*/
//__inline__ __device__ double atan2(double a,double b)
//{
//#if 0
//	return 0;
//#else
//	/*return atan2(a,b);*/
//#endif
//}
/*------------------------------------------------------------------------------------------*/
/**
*
*
*
*/
/*------------------------------------------------------------------------------------------*/
//__inline__ __device__ double fabs(double x)
//{
//#if 0
//	return 0;
//#else
//	/*return fabs(x);*/
//#endif
//}
/*------------------------------------------------------------------------------------------*/
/**
*
*
*
*/
/*------------------------------------------------------------------------------------------*/
struct Lock {
	int *mutex;
	Lock( void ) {
		( hipMalloc( (void**)&mutex, sizeof(int) ) );
		( hipMemset( mutex, 0, sizeof(int) ) );
	}
	~Lock( void ) {
		hipFree( mutex );
	}
	__device__ void lock( void ) {
		while( atomicCAS( mutex, 0, 1 ) != 0 );
	}
	__device__ void unlock( void ) {
		atomicExch( mutex, 0 );
	}
};
/*------------------------------------------------------------------------------------------*/
/**
*
*cuda��ʼ��
*@retval true ��ʼ���ɹ�
*@retval false ��ʼ��ʧ��
*/
/*------------------------------------------------------------------------------------------*/
bool cudaInit_CUI(void)
{
		 int i;
		 int device_count;

		 if( hipGetDeviceCount(&device_count) )
		 {
			 printf(" There is zero device beyond 1.0\n"); 
			 return false;
		 }
#if _DEBUG
		//printf("CUDA DEVICE NUMS: %d \n",device_count);
#endif
		 // �ҵ�һ�����õ��豸
		 for(i=0;i<device_count;i++)
		 {
			  struct hipDeviceProp_t device_prop;
			  if(hipGetDeviceProperties(&device_prop,i)==hipSuccess)
			  {
#if 0
	printf("device properties is :\n"
			"\t device name is %s\n"
			"\t totalGlobalMem is %d\n"
			"\t sharedMemPerBlock is %d\n"
			"\t regsPerBlock is %d\n"
			"\t warpSize is %d\n"
			"\t memPitch is %d\n"
			"\t maxThreadsPerBlock is %d\n"
			"\t maxThreadsDim [3] is %d X %d X %d\n"
			"\t maxGridSize [3] is %d X %d X %d\n"
			"\t totalConstMem is %d\n"
			"\t device version is major %d ,minor %d\n"
			"\t clockRate is %d\n"
			"\t textureAlignment is %d\n"
			"\t deviceOverlap is %d\n"
			"\t multiProcessorCount is %d\n",
			device_prop.name,
			device_prop.totalGlobalMem,
			device_prop.sharedMemPerBlock,
			device_prop.regsPerBlock,
			device_prop.warpSize,
			device_prop.memPitch,
			device_prop.maxThreadsPerBlock,
			device_prop.maxThreadsDim[0],device_prop.maxThreadsDim[1],device_prop.maxThreadsDim[2],
			device_prop.maxGridSize[0],device_prop.maxGridSize[1],device_prop.maxGridSize[2],
			device_prop.totalConstMem,
			device_prop.major,device_prop.minor,
			device_prop.clockRate,
			device_prop.textureAlignment,
			device_prop.deviceOverlap,
			device_prop.multiProcessorCount);
#endif				
					/* if(hipSetDevice(i)==hipSuccess){
						 printf("USE GPU ID: %d \n",i);
						 return true;
					 }*/
				break;
			  }

		 }
		 
		 if(i==device_count)
		 {
			  printf("Get the propertites of device occurred error\n");
			  return false;
		 }
		 //�п����豸
		 int default_id,expect_id;		 
		 int thread_id=GetThreadIdSelfwinlinux();
		 expect_id=thread_id%device_count;
		
		 printf("CUDA NUMS: %d, ID: %d,EXPECT: %d \n",device_count,thread_id,expect_id);

		 hipGetDevice(&default_id);

		 if(default_id==expect_id){
			 return true;
		 }else{			 
			 //�豸����
			 if(hipSetDevice(expect_id)==hipSuccess){
				 //printf("USE GPU ID: %d \n",expect_id);
				 return true;
			 }
		 }
		 
		 //////////////////////////////////////////
	//	 int ket=kernelExecTimeoutEnabled();
		 /////////////////////////////////////////
		 return false;
	
}
/*------------------------------------------------------------------------------------------*/
/**
*
*cudaȥ��ʼ��
*@return 
*@retval true ��ʼ���ɹ�
*@retval false ��ʼ��ʧ��
*/
/*------------------------------------------------------------------------------------------*/
bool cudaDeInit_CUI(void)
{
#if 0
	hipError_t cudaStatus;
	cudaStatus=hipDeviceReset();
	if(cudaStatus!=hipSuccess){
		return false;
	}else{
		return true;
	}
#else
	return true;
#endif
	
}
/*------------------------------------------------------------------------------------------*/
/**
*Cuda�ȴ���ʱ��ȡ�������
*
*
*/
/*------------------------------------------------------------------------------------------*/
void cudaGetLastError_Sync_CUI(void)
{
	hipError_t cudaStatus_syn,cudaStatus;
	const char  *errstr=NULL;


	do 
	{
		cudaStatus_syn=hipDeviceSynchronize();
		if (cudaStatus_syn==hipSuccess)
		{
			break;

		}else{
			cudaStatus=hipGetLastError();
			if (cudaStatus!=hipSuccess){
				errstr=hipGetErrorString(cudaStatus);
				printf("hipDeviceSynchronize: %s",errstr);
			}
		}
		


	} while (cudaStatus_syn!=hipSuccess);

}
/*------------------------------------------------------------------------------------------*/
/**
*
*ת��LABɫ�ʿռ�ɽǶȡ�ģ�������ȱ�ʾ��
*
*@param m_width ͼ����
*@param m_height ͼ��߶�
*@param L ���ص��L����
*@param A ���ص��A����
*@param B ���ص��B����
*@param X ���ص��X����
*@param Y ���ص��Y����
*@param sita_n ���ص�ĽǶ� ����һ����[0,1]��
*@param m_n    ���ص��ģ�� ����һ����[0,1]��
*@param L_n    ���ص������ ����һ����[0,1]��
*@param X_n    ���ص��X���꣨��һ����[0,1]��
*@param Y_n    ���ص��Y���꣨��һ����[0,1]��
*
*/
/*------------------------------------------------------------------------------------------*/
__inline__ __host__ __device__ void ConvertLab2oml(
	int m_width,
	int m_height,
	double L,
	double A,
	double B,
	double X,
	double Y,
	double* sita_n,
	double* m_n,
	double* L_n,
	double* X_n,
	double* Y_n)
{
	assert(L>=0&&L<=100+1);
	assert(A>=-128&&A<=128+1);
	assert(B>=-128&&B<=128+1);
	assert(X>=0&&X<=m_width+1);
	assert(Y>=0&&Y<=m_height+1);
#if 1
	///////////////////////////////////////
	*sita_n=atan2(A,B);
	*sita_n=*sita_n/(2*3.1415927)+0.5;
	///////////////////////////////////////
	*m_n=sqrt(A*A+B*B);
	*m_n=*m_n/(128*1.415);
	///////////////////////////////////////
	*L_n=L/100;
	assert(*L_n>=0&&*L_n<=1+1E-1);
	///////////////////////////////////////
	*X_n=X/m_width;
	assert(*X_n>=0&&*X_n<=1+1E-1);
	*Y_n=Y/m_height;
	assert(*Y_n>=0&&*Y_n<=1+1E-1);
#endif
	assert(*sita_n>=0-0.1&&*sita_n<=1+0.1);
	assert(*m_n>=0-0.1&&*m_n<=1+0.1);
	assert(*L_n>=0-0.1&&*L_n<=1+0.1);
	assert(*X_n>=0-0.1&&*X_n<=1+0.1);
	assert(*Y_n>=0-0.1&&*Y_n<=1+0.1);
}
/*------------------------------------------------------------------------------------------*/
/**
*
*�����������ص�ľ���
*@image html image027.png "ɫ�ʹ�ʽ"
*@image html image029.png "���빫ʽ"
*@image html image031.png "�����ۺϹ�ʽ"
*
*
*@param  alpha     �Ƕ�ϵ��
*@param  betta     ģ��ϵ��
*@param  gama      ����ϵ��
*@param  fai       ����ϵ��
*@param  sita_n0   ����0�ĽǶ�
*@param  m_n0      ����0��ģ��
*@param  L_n0      ����0������
*@param  X_n0      ����0�ĵȼ�X����
*@param  Y_n0      ����0�ĵȼ�Y����
*@param  sita_n1   ����1�ĽǶ�
*@param  m_n1      ����1��ģ��
*@param  L_n1      ����1������
*@param  X_n1      ����1�ĵȼ�X����
*@param  Y_n1      ����1�ĵȼ�Y����
*
*
*
*/
/*------------------------------------------------------------------------------------------*/
__inline__ __device__ double  CalculateNewDistance(
	float alpha,
	float betta,
	float gama,
	float fai,
	double sita_n0,
	double m_n0,
	double L_n0,
	double X_n0,
	double Y_n0,
	double sita_n1,
	double m_n1,
	double L_n1,
	double X_n1,
	double Y_n1)
{
	/*alpha=0.6;
	betta=0.3;
	gama=1-alpha-betta;*/
	/*float fai=100;*/
	assert(sita_n0>=0-0.1&&sita_n0<=1+0.1);
	assert(L_n0>=0-0.1&&L_n0<=1+0.1);
	assert(X_n0>=0-0.1&&X_n0<=1+0.1);
	assert(Y_n0>=0-0.1&&Y_n0<=1+0.1);
	assert(sita_n1>=0-0.1&&sita_n1<=1+0.1);
    assert(L_n1>=0-0.1&&L_n1<=1+0.1);
	assert(X_n1>=0-0.1&&X_n1<=1+0.1);
	assert(Y_n1>=0-0.1&&Y_n1<=1+0.1);
#if 0
	double dst=alpha*fabs(sita_n1-sita_n0)+betta*fabs(m_n1-m_n0)+gama*fabs(L_n1-L_n0);
	double dst_xy=fai*sqrt((X_n1-X_n0)*(X_n1-X_n0)+(Y_n1-Y_n0)*(Y_n1-Y_n0));
	dst+=dst_xy;
	return dst;
#else
	double dst=alpha*fabs(sita_n1-sita_n0)+betta*fabs(m_n1-m_n0)+gama*fabs(L_n1-L_n0);
	double dst_xy=fai*(fabs(X_n1-X_n0)+fabs(Y_n1-Y_n0));
	dst+=dst_xy;
	return dst;
#endif
	
}
/*------------------------------------------------------------------------------------------*/
/**
*һ���������������Ӽ�����뱣����Сֵ
*
*@param     kseedsSize ���Ӹ��� 
*@param     alpha      �Ƕ�ϵ��
*@param     betta      ģ��ϵ��
*@param     gama       ����ϵ��
*@param     fai        �������
*@param     m_width    ͼ����
*@param     m_height   ͼ��߶�
*@param   [in]  m_lvec     ͼ��L����
*@param   [in]  m_avec     ͼ��A����
*@param   [in] m_bvec     ͼ��B����
*@param   [in]  dev_sita_n  ͼ��Ƕȷ���
*@param   [in] dev_m_n    ͼ��ģ������
*@param   [in]  dev_L_n   ͼ�����ȷ���
*@param   [in] dev_X_n   ͼ��X����
*@param   [in]  dev_Y_n   ͼ��Y����
*@param   [in]  kseedsl   ���ӵ�L����
*@param   [in] kseedsa   ���ӵ�A����
*@param   [in] kseedsb   ���ӵ�B����
*@param   [in] kseedsx   ���ӵ�X�������
*@param   [in] kseedsy   ���ӵ�Y�������
*@param   [in] dev_kseed_sita_n  ���ӽǶȷ���
*@param   [in] dev_kseed_m_n     ����ģ������
*@param   [in] dev_kseed_L_n     ���ӵ�L����
*@param   [in] dev_kseed_X_n     ���ӵ�X����
*@param   [in] dev_kseed_Y_n     ���ӵ�Y����
*@param   [in] klabels           ͼ��ı궨��Ϣ
*@param   [in] distvec           δʹ��
*
*
*/
/*------------------------------------------------------------------------------------------*/
__global__ void SetMinimumOnLab(
	const int kseedsSize,
	float alpha,
	float betta,
	float gama,
	float fai,
	const int m_width,
	const int m_height,
	double* m_lvec,
	double* m_avec,
	double* m_bvec,
	double* kseedsl,
	double* kseedsa,
	double* kseedsb,
	double* kseedsx,
	double* kseedsy,
	int*   klabels,
	double* distvec,
	const int			STEP,
	const double		M)
{

	int x1, y1, x2, y2;
	double l, a, b;
	double dist;
	//double distxy;
	int SpIdx = threadIdx.x+blockIdx.x*blockDim.x;
	int n=SpIdx;
	int offset = STEP;
	if (SpIdx<kseedsSize){
#if 1
///////////////////////////////////////////////
		y1 = max(0.0,			kseedsy[n]-offset);
		y2 = min((double)m_height,	kseedsy[n]+offset);
		x1 = max(0.0,			kseedsx[n]-offset);
		x2 = min((double)m_width,	kseedsx[n]+offset);
	//////////////////////////////////////////////
		for( int y = y1; y < y2; y++ )
			{
				for( int x = x1; x < x2; x++ )
				{
					int i = y*m_width + x;

					l = m_lvec[i];
					a = m_avec[i];
					b = m_bvec[i];
#if 0
					dist =			(l - kseedsl[n])*(l - kseedsl[n]) +
						(a - kseedsa[n])*(a - kseedsa[n]) +
						(b - kseedsb[n])*(b - kseedsb[n]);
					//if (itr>5)
					{
						distxy =		(x - kseedsx[n])*(x - kseedsx[n]) +
							(y - kseedsy[n])*(y - kseedsy[n]);
					}
					//else
					//{
					//	distxy =0;
					//}
					//------------------------------------------------------------------------
					dist += distxy*invwt;//dist = sqrt(dist) + sqrt(distxy*invwt);//this is more exact
					//------------------------------------------------------------------------
#endif
#if 1
					double sita_n, m_n, L_n, X_n, Y_n;
					double Ksita_n, Km_n, KL_n, KX_n, KY_n;
					ConvertLab2oml(m_width,m_height,l,a,b,x,y,&sita_n,&m_n,&L_n,&X_n,&Y_n);
					ConvertLab2oml(m_width,m_height,kseedsl[n],kseedsa[n],kseedsb[n],kseedsx[n],kseedsy[n],&Ksita_n,&Km_n,&KL_n,&KX_n,&KY_n);
					dist=CalculateNewDistance(alpha,betta,gama,fai,sita_n,m_n,L_n,X_n,Y_n,Ksita_n,Km_n,KL_n,KX_n,KY_n);
#endif
					if( dist < distvec[i] )
					{
						distvec[i] = dist;
						klabels[i]  = n;
						//atomicExch(&klabels[i],n);
					}
				}
			}

#endif
	}
}
/*------------------------------------------------------------------------------------------*/
/**
*���ó�����ͼƬ��Labels
*
*@param   x_offset   Χ����������X��ƫ����
*@param   y_offset   Χ����������Y��ƫ����
*@param   offsetSize ƫ������ĳ���
*@param   alpha      �Ƕ�ϵ��
*@param   betta      ģ��ϵ��
*@param   gama       ����ϵ��
*@param   fai        ����ϵ��
*@param   SeedWH     ����ƽ����������
*@param   labelsX    �����ؿ�Ŀ��
*@param   labelsY    �����ؿ�ĳ���
*@param   x          ����X����
*@param   y          ����Y����
*@param   m_width    ͼ����
*@param   m_height   ͼ��߶�
*@param   dev_kseeds_sita_n  ͼ��ĽǶȾ���
*@param   dev_kseeds_m_n     ͼ��Ŀ�Ⱦ���
*@param   dev_kseeds_L_n     ͼ������Ⱦ���
*@param   dev_kseeds_X_n     ͼ��ĵȼ�X����
*@param   dev_kseeds_Y_n     ͼ��ĵȼ�Y����
*@param   dev_sita_n         ͼ��ĽǶȷ�������
*@param   dev_m_n			 ͼ���ģ����������
*@param   dev_L_n            ͼ������ȷ�������
*@param   dev_X_n            ͼ���X��������
*@param   dev_Y_n            ͼ���Y��������
*@param   klabels            ͼ��ı궨��Ϣ
*@param   distvec            ���������Сֵ�ľ���
*
*/
/*------------------------------------------------------------------------------------------*/
__inline__ __device__  void SetPixelLables(
	const int* x_offset,
	const int* y_offset,
	const int offsetSize,
	const float alpha,
	const float betta,
	const float gama,
	const float fai,
	const int SeedWH,
	const int labelsX,
	const int labelsY,
	const int x,
	const int y,
	const int m_width,
	const int m_height,
	const double* dev_kseeds_sita_n,
	const double* dev_kseeds_m_n,
	const double* dev_kseeds_L_n,
	const double* dev_kseeds_X_n,
	const double* dev_kseeds_Y_n,
	const double* dev_sita_n,
	const double* dev_m_n,
	const double* dev_L_n,
	const double* dev_X_n,
	const double* dev_Y_n,
	int*   klabels,
	double* distvec)
{
#if 0
	{
		double dist;
		//////////////////////////////////////////////
		int i = y*m_width + x;
		////////////////////////////////////////////
		for (int xi=0;xi<offsetSize;xi++){
			for (int yi=0;yi<offsetSize;yi++){
				int nx=labelsX+x_offset[xi];
				int ny=labelsY+y_offset[yi];
				if (nx>=0&&nx<SeedWH&&ny>=0&&ny<SeedWH){
					int ni=nx+ny*SeedWH;
					{
						dist=CalculateNewDistance(alpha,betta,gama,fai,
							dev_sita_n[i],dev_m_n[i],dev_L_n[i],dev_X_n[i],dev_Y_n[i],
							dev_kseeds_sita_n[ni],dev_kseeds_m_n[ni],dev_kseeds_L_n[ni],dev_kseeds_X_n[ni],dev_kseeds_Y_n[ni]);
						if( dist < distvec[i] )
						{
							distvec[i] = dist;
							klabels[i]  = ni;
						}

					}
				}
			}
		}
	}
#endif
#if 1
	{
		double dist;
		//////////////////////////////////////////////
		int i = y*m_width + x;
		////////////////////////////////////////////
		for (int xyi=0;xyi<offsetSize;xyi++){			
				int nx=labelsX+x_offset[xyi];
				int ny=labelsY+y_offset[xyi];
				if (nx>=0&&nx<SeedWH&&ny>=0&&ny<SeedWH){
					int ni=nx+ny*SeedWH;
					{
						dist=CalculateNewDistance(alpha,betta,gama,fai,
							dev_sita_n[i],dev_m_n[i],dev_L_n[i],dev_X_n[i],dev_Y_n[i],
							dev_kseeds_sita_n[ni],
							dev_kseeds_m_n[ni],
							dev_kseeds_L_n[ni],
							dev_kseeds_X_n[ni],
							dev_kseeds_Y_n[ni]);

						if( dist < distvec[i] )
						{
							distvec[i] = dist;
							klabels[i]  = ni;
						}

					}
				}
			
		}
	}
#endif
}
/*------------------------------------------------------------------------------------------*/
/**
*
*һ��������5�����Ӽ�����뱣����Сֵ
*
*@param     kseedsSize ���Ӹ��� 
*@param     alpha      �Ƕ�ϵ��
*@param     betta      ģ��ϵ��
*@param     gama       ����ϵ��
*@param     fai        �������
*@param     m_width    ͼ����
*@param     m_height   ͼ��߶�
*@param   [in]  m_lvec     ͼ��L����
*@param   [in]  m_avec     ͼ��A����
*@param   [in] m_bvec     ͼ��B����
*@param   [in]  dev_sita_n  ͼ��Ƕȷ���
*@param   [in] dev_m_n    ͼ��ģ������
*@param   [in]  dev_L_n   ͼ�����ȷ���
*@param   [in] dev_X_n   ͼ��X����
*@param   [in]  dev_Y_n   ͼ��Y����
*@param   [in]  kseedsl   ���ӵ�L����
*@param   [in] kseedsa   ���ӵ�A����
*@param   [in] kseedsb   ���ӵ�B����
*@param   [in] kseedsx   ���ӵ�X�������
*@param   [in] kseedsy   ���ӵ�Y�������
*@param   [in] dev_kseed_sita_n  ���ӽǶȷ���
*@param   [in] dev_kseed_m_n     ����ģ������
*@param   [in] dev_kseed_L_n     ���ӵ�L����
*@param   [in] dev_kseed_X_n     ���ӵ�X����
*@param   [in] dev_kseed_Y_n     ���ӵ�Y����
*@param   [in] klabels           ͼ��ı궨��Ϣ
*@param   [in] distvec           δʹ��
*
*/
/*------------------------------------------------------------------------------------------*/
__global__ void SetMinimumOnLab_5Seeds(
	const int kseedsSize,
	float alpha,
	float betta,
	float gama,
	float fai,
	const int m_width,
	const int m_height,
	double* m_lvec,
	double* m_avec,
	double* m_bvec,
	double* dev_sita_n,
	double* dev_m_n,
	double* dev_L_n,
	double* dev_X_n,
	double* dev_Y_n,
	double* kseedsl,
	double* kseedsa,
	double* kseedsb,
	double* kseedsx,
	double* kseedsy,
	double*	dev_kseed_sita_n,
	double*	dev_kseed_m_n,
	double*	dev_kseed_L_n,
	double*	dev_kseed_X_n,
	double*	dev_kseed_Y_n,
	int*   klabels,
	double* distvec)
{
	int x_offset[]={0,1,-1,0,0};
	int y_offset[]={0,0,0,1,-1};
////////////////////////////////////////////////////////////////////////
#if 1
{
	int x =blockIdx.x*blockDim.x+threadIdx.x;
	int y =blockIdx.y*blockDim.y+threadIdx.y;
	int SeedWH=sqrt((float)kseedsSize);	
	/////////////////////////////////////////////
	int offsetSize=sizeof(x_offset)/sizeof(int);
	if (x>=0&&x<m_width&&y>=0&&y<m_height){
		int ind=x+y*m_width;
		int labels_ind=klabels[ind];
	assert(labels_ind>=-1&&labels_ind<kseedsSize);
#if 1
		if (labels_ind==-1){
			int x_step=m_width/SeedWH;
			int y_step=m_height/SeedWH;
			int labelsX=x/x_step;
			int labelsY=y/y_step;
			SetPixelLables(x_offset,y_offset,offsetSize,
				alpha,betta,gama,fai,
				SeedWH,labelsX,labelsY,x,y,m_width,m_height,
				dev_kseed_sita_n,dev_kseed_m_n,dev_kseed_L_n,dev_kseed_X_n,dev_kseed_Y_n,
				dev_sita_n,dev_m_n,dev_L_n,dev_X_n,dev_Y_n,				
				klabels,distvec);
		}else{
			int labelsX=labels_ind%SeedWH;
			int labelsY=labels_ind/SeedWH;
#if 1
			SetPixelLables(x_offset,y_offset,offsetSize,
				alpha,betta,gama,fai,
				SeedWH,labelsX,labelsY,x,y,m_width,m_height,
				dev_kseed_sita_n,dev_kseed_m_n,dev_kseed_L_n,dev_kseed_X_n,dev_kseed_Y_n,
				dev_sita_n,dev_m_n,dev_L_n,dev_X_n,dev_Y_n,
				klabels,distvec);
#endif	
		}
#endif
	}

}
#endif
////////////////////////////////////////////////////////////////////////
}
/*------------------------------------------------------------------------------------------*/
/**
*
*һ��������9�����Ӽ�����뱣����Сֵ
*
*@param     kseedsSize ���Ӹ��� 
*@param     alpha      �Ƕ�ϵ��
*@param     betta      ģ��ϵ��
*@param     gama       ����ϵ��
*@param     fai        �������
*@param     m_width    ͼ����
*@param     m_height   ͼ��߶�
*@param   [in]  m_lvec     ͼ��L����
*@param   [in]  m_avec     ͼ��A����
*@param   [in] m_bvec     ͼ��B����
*@param   [in]  dev_sita_n  ͼ��Ƕȷ���
*@param   [in] dev_m_n    ͼ��ģ������
*@param   [in]  dev_L_n   ͼ�����ȷ���
*@param   [in] dev_X_n   ͼ��X����
*@param   [in]  dev_Y_n   ͼ��Y����
*@param   [in]  kseedsl   ���ӵ�L����
*@param   [in] kseedsa   ���ӵ�A����
*@param   [in] kseedsb   ���ӵ�B����
*@param   [in] kseedsx   ���ӵ�X�������
*@param   [in] kseedsy   ���ӵ�Y�������
*@param   [in] dev_kseed_sita_n  ���ӽǶȷ���
*@param   [in] dev_kseed_m_n     ����ģ������
*@param   [in] dev_kseed_L_n     ���ӵ�L����
*@param   [in] dev_kseed_X_n     ���ӵ�X����
*@param   [in] dev_kseed_Y_n     ���ӵ�Y����
*@param   [in] klabels           ͼ��ı궨��Ϣ
*@param   [in] distvec           δʹ��
*
*/
/*------------------------------------------------------------------------------------------*/
__global__ void SetMinimumOnLab_9Seeds(
	const int kseedsSize,
	float alpha,
	float betta,
	float gama,
	float fai,
	const int m_width,
	const int m_height,
	double* m_lvec,
	double* m_avec,
	double* m_bvec,
	double* dev_sita_n,
	double* dev_m_n,
	double* dev_L_n,
	double* dev_X_n,
	double* dev_Y_n,
	double* kseedsl,
	double* kseedsa,
	double* kseedsb,
	double* kseedsx,
	double* kseedsy,
	double*	dev_kseed_sita_n,
	double*	dev_kseed_m_n,
	double*	dev_kseed_L_n,
	double*	dev_kseed_X_n,
	double*	dev_kseed_Y_n,
	int*   klabels,
	double* distvec)
{
////////////////////////////////////////////////////////////////////////
	int x_offset[]={0,1,-1,0,0,1,-1,1,-1};
	int y_offset[]={0,0,0,1,-1,1,-1,-1,1};
#if 1
	{
		int x =blockIdx.x*blockDim.x+threadIdx.x;
		int y =blockIdx.y*blockDim.y+threadIdx.y;
		int SeedWH=sqrt((float)kseedsSize);	

		int offsetSize=sizeof(x_offset)/sizeof(int);
		if (x>=0&&x<m_width&&y>=0&&y<m_height){
			int ind=x+y*m_width;
			int labels_ind=klabels[ind];
			assert(labels_ind>=-1&&labels_ind<kseedsSize);
#if 1
			if (labels_ind==-1){
				float x_step=1.0*m_width/SeedWH;
				float y_step=1.0*m_height/SeedWH;
				int labelsX=x/x_step;
				int labelsY=y/y_step;
				SetPixelLables(x_offset,y_offset,offsetSize,
					alpha,betta,gama,fai,
					SeedWH,labelsX,labelsY,x,y,m_width,m_height,
					dev_kseed_sita_n,dev_kseed_m_n,dev_kseed_L_n,dev_kseed_X_n,dev_kseed_Y_n,
					dev_sita_n,dev_m_n,dev_L_n,dev_X_n,dev_Y_n,				
					klabels,distvec);
			}else{
				int labelsX=labels_ind%SeedWH;
				int labelsY=labels_ind/SeedWH;
#if 1
				SetPixelLables(x_offset,y_offset,offsetSize,
					alpha,betta,gama,fai,
					SeedWH,labelsX,labelsY,x,y,m_width,m_height,
					dev_kseed_sita_n,dev_kseed_m_n,dev_kseed_L_n,dev_kseed_X_n,dev_kseed_Y_n,
					dev_sita_n,dev_m_n,dev_L_n,dev_X_n,dev_Y_n,
					klabels,distvec);
#endif

			}
#endif
		}

	}
#endif
////////////////////////////////////////////////////////////////////////
}
/*------------------------------------------------------------------------------------------*/
/**
*
*һ��������9�����Ӽ�����뱣����Сֵ
*
*@param     kseedsSize ���Ӹ��� 
*@param     alpha      �Ƕ�ϵ��
*@param     betta      ģ��ϵ��
*@param     gama       ����ϵ��
*@param     fai        �������
*@param     m_width    ͼ����
*@param     m_height   ͼ��߶�
*@param   [in]  m_lvec     ͼ��L����
*@param   [in]  m_avec     ͼ��A����
*@param   [in] m_bvec     ͼ��B����
*@param   [in]  dev_sita_n  ͼ��Ƕȷ���
*@param   [in] dev_m_n    ͼ��ģ������
*@param   [in]  dev_L_n   ͼ�����ȷ���
*@param   [in] dev_X_n   ͼ��X����
*@param   [in]  dev_Y_n   ͼ��Y����
*@param   [in]  kseedsl   ���ӵ�L����
*@param   [in] kseedsa   ���ӵ�A����
*@param   [in] kseedsb   ���ӵ�B����
*@param   [in] kseedsx   ���ӵ�X�������
*@param   [in] kseedsy   ���ӵ�Y�������
*@param   [in] dev_kseed_sita_n  ���ӽǶȷ���
*@param   [in] dev_kseed_m_n     ����ģ������
*@param   [in] dev_kseed_L_n     ���ӵ�L����
*@param   [in] dev_kseed_X_n     ���ӵ�X����
*@param   [in] dev_kseed_Y_n     ���ӵ�Y����
*@param   [in] klabels           ͼ��ı궨��Ϣ
*@param   [in] distvec           δʹ��
*
*/
/*------------------------------------------------------------------------------------------*/
__global__ void SetMinimumOnThetaMLXY_9Seeds(
	const int kseedsSize,
	float alpha,
	float betta,
	float gama,
	float fai,
	const int m_width,
	const int m_height,
	double* dev_sita_n,
	double* dev_m_n,
	double* dev_L_n,
	double* dev_X_n,
	double* dev_Y_n,
	double*	dev_kseed_sita_n,
	double*	dev_kseed_m_n,
	double*	dev_kseed_L_n,
	double*	dev_kseed_X_n,
	double*	dev_kseed_Y_n,
	int*   klabels,
	double* distvec)
{
////////////////////////////////////////////////////////////////////////
	int x_offset[]={0,1,-1,0,0,1,-1,1,-1};
	int y_offset[]={0,0,0,1,-1,1,-1,-1,1};
#if 1
	{
		int x =blockIdx.x*blockDim.x+threadIdx.x;
		int y =blockIdx.y*blockDim.y+threadIdx.y;
		int SeedWH=sqrt((float)kseedsSize);	

		int offsetSize=sizeof(x_offset)/sizeof(int);
		if (x>=0&&x<m_width&&y>=0&&y<m_height){
			int ind=x+y*m_width;
			int labels_ind=klabels[ind];
			assert(labels_ind>=-1&&labels_ind<kseedsSize);
#if 1
			if (labels_ind==-1){
				float x_step=1.0*m_width/SeedWH;
				float y_step=1.0*m_height/SeedWH;
				int labelsX=x/x_step;
				int labelsY=y/y_step;
				SetPixelLables(x_offset,y_offset,offsetSize,
					alpha,betta,gama,fai,
					SeedWH,labelsX,labelsY,x,y,m_width,m_height,
					dev_kseed_sita_n,dev_kseed_m_n,dev_kseed_L_n,dev_kseed_X_n,dev_kseed_Y_n,
					dev_sita_n,dev_m_n,dev_L_n,dev_X_n,dev_Y_n,				
					klabels,distvec);
			}else{
				int labelsX=labels_ind%SeedWH;
				int labelsY=labels_ind/SeedWH;
#if 1
				SetPixelLables(x_offset,y_offset,offsetSize,
					alpha,betta,gama,fai,
					SeedWH,labelsX,labelsY,x,y,m_width,m_height,
					dev_kseed_sita_n,dev_kseed_m_n,dev_kseed_L_n,dev_kseed_X_n,dev_kseed_Y_n,
					dev_sita_n,dev_m_n,dev_L_n,dev_X_n,dev_Y_n,
					klabels,distvec);
#endif

			}
#endif
		}

	}
#endif
////////////////////////////////////////////////////////////////////////
}
/*------------------------------------------------------------------------------------------*/
/**
*
*
*
*/
/*------------------------------------------------------------------------------------------*/
__global__ void SetMinimumOnLab_AllSeeds(
	const int kseedsSize,
	float alpha,
	float betta,
	float gama,
	float fai,
	const int m_width,
	const int m_height,
	double* m_lvec,
	double* m_avec,
	double* m_bvec,
	double* dev_sita_n,
	double* dev_m_n,
	double* dev_L_n,
	double* dev_X_n,
	double* dev_Y_n,
	double* kseedsl,
	double* kseedsa,
	double* kseedsb,
	double* kseedsx,
	double* kseedsy,
	double*	dev_kseed_sita_n,
	double*	dev_kseed_m_n,
	double*	dev_kseed_L_n,
	double*	dev_kseed_X_n,
	double*	dev_kseed_Y_n,
	int*   klabels,
	double* distvec)
{
	////////////////////////////////////////////////////////////////////////
#if 0
	int x =blockIdx.x*blockDim.x+threadIdx.x;
	int y =blockIdx.y*blockDim.y+threadIdx.y;
	int ind=x+y*m_width;
	int labels_ind;
	if (x>=0&&x<m_width&&y>=0&&y<m_height){
		labels_ind= klabels[ind];	
		assert(labels_ind>=-1&&labels_ind<kseedsSize);
		{
			double l, a, b;
			double dist;
			//////////////////////////////////////////////
			int i = y*m_width + x;
			l = m_lvec[i];
			a = m_avec[i];
			b = m_bvec[i];
			double sita_n, m_n, L_n, X_n, Y_n;
			double Ksita_n, Km_n, KL_n, KX_n, KY_n;
			ConvertLab2oml(m_width,m_height,l,a,b,x,y,sita_n,m_n,L_n,X_n,Y_n);
			int kseedareaXmin=max(0,x-STEP);
			int kseedareaYmin=max(0,y-STEP);
			int kseedareaXmax=min(m_width,x+STEP);
			int kseedareaYmax=min(m_height,y+STEP);

			for(int ni=0;ni<kseedsSize;ni++){
				int n=ni;
				int KseedX=kseedsx[n];
				int KseedY=kseedsy[n];
				if ((KseedX>=kseedareaXmin)&&
					(KseedX<kseedareaXmax)&&
					(KseedY>=kseedareaYmin)&&
					(KseedY<kseedareaYmax)){
						ConvertLab2oml(m_width,m_height,kseedsl[n],kseedsa[n],kseedsb[n],kseedsx[n],kseedsy[n],Ksita_n,Km_n,KL_n,KX_n,KY_n);
						dist=CalculateNewDistance(alpha,betta,gama,fai,sita_n,m_n,L_n,X_n,Y_n,Ksita_n,Km_n,KL_n,KX_n,KY_n);
						if( dist < distvec[i] )
						{
							distvec[i] = dist;
							klabels[i]  = n;
						}
				}

			}

		}

	}

#endif
	////////////////////////////////////////////////////////////////////////
#if 0
	{
		int x =blockIdx.x*blockDim.x+threadIdx.x;
		int y =blockIdx.y*blockDim.y+threadIdx.y;
		int ind=x+y*m_width;
		int labels_ind;
		if (x>=0&&x<m_width&&y>=0&&y<m_height){
			labels_ind= klabels[ind];	
			assert(labels_ind>=-1&&labels_ind<kseedsSize);
			{
				double l, a, b;
				double dist;
				double sita_n, m_n, L_n, X_n, Y_n;
				double Ksita_n, Km_n, KL_n, KX_n, KY_n;
				//////////////////////////////////////////////
				int i = y*m_width + x;
				////////////////////////////////////////////
				sita_n=dev_sita_n[i];
				m_n=dev_m_n[i];
				L_n=dev_L_n[i];
				X_n=dev_X_n[i];
				Y_n=dev_Y_n[i];
				int kseedareaXmin=max(0,x-STEP);
				int kseedareaYmin=max(0,y-STEP);
				int kseedareaXmax=min(m_width,x+STEP);
				int kseedareaYmax=min(m_height,y+STEP);

				for(int ni=0;ni<kseedsSize;ni++){
					int KseedX=kseedsx[ni];
					int KseedY=kseedsy[ni];
					if ((KseedX>=kseedareaXmin)&&
						(KseedX<kseedareaXmax)&&
						(KseedY>=kseedareaYmin)&&
						(KseedY<kseedareaYmax)){
							Ksita_n=dev_sita_n[ni];
							Km_n=dev_m_n[ni];
							KL_n=dev_L_n[ni];
							KX_n=dev_X_n[ni];
							KY_n=dev_Y_n[ni];
							dist=CalculateNewDistance(alpha,betta,gama,fai,sita_n,m_n,L_n,X_n,Y_n,Ksita_n,Km_n,KL_n,KX_n,KY_n);
							if( dist < distvec[i] )
							{
								distvec[i] = dist;
								klabels[i]  = ni;
							}
					}

				}

			}

		}
	}
#endif
	////////////////////////////////////////////////////////////////////////
#if 1
	{
		int x =blockIdx.x*blockDim.x+threadIdx.x;
		int y =blockIdx.y*blockDim.y+threadIdx.y;
		int SeedWH=sqrt((float)kseedsSize);	
		int x_step=m_width/SeedWH;
		int y_step=m_height/SeedWH;
		int STEP=x_step+y_step;
		int ind=x+y*m_width;
		int labels_ind=klabels[ind];
		if (labels_ind==-1){
#if 1
			if (x>=0&&x<m_width&&y>=0&&y<m_height){
				labels_ind= klabels[ind];	
				assert(labels_ind>=-1&&labels_ind<kseedsSize);
				{
					double l, a, b;
					double dist;
					double sita_n, m_n, L_n, X_n, Y_n;
					double Ksita_n, Km_n, KL_n, KX_n, KY_n;
					//////////////////////////////////////////////
					int i = y*m_width + x;
					////////////////////////////////////////////
					sita_n=dev_sita_n[i];
					m_n=dev_m_n[i];
					L_n=dev_L_n[i];
					X_n=dev_X_n[i];
					Y_n=dev_Y_n[i];
					int kseedareaXmin=max(0,x-STEP);
					int kseedareaYmin=max(0,y-STEP);
					int kseedareaXmax=min(m_width,x+STEP);
					int kseedareaYmax=min(m_height,y+STEP);

					for(int ni=0;ni<kseedsSize;ni++){
						int KseedX=kseedsx[ni];
						int KseedY=kseedsy[ni];
						if ((KseedX>=kseedareaXmin)&&
							(KseedX<kseedareaXmax)&&
							(KseedY>=kseedareaYmin)&&
							(KseedY<kseedareaYmax)){
								Ksita_n=dev_sita_n[ni];
								Km_n=dev_m_n[ni];
								KL_n=dev_L_n[ni];
								KX_n=dev_X_n[ni];
								KY_n=dev_Y_n[ni];
								dist=CalculateNewDistance(alpha,betta,gama,fai,sita_n,m_n,L_n,X_n,Y_n,Ksita_n,Km_n,KL_n,KX_n,KY_n);
								if( dist < distvec[i] )
								{
									distvec[i] = dist;
									klabels[i]  = ni;
								}
						}
					}
				}
			}
#endif
		}else{
#if 1
			{
				int SeedWH=sqrt((float)kseedsSize);
				int labelsX=labels_ind%SeedWH;
				int labelsY=labels_ind/SeedWH;
				int x_offset[]={0,1,-1,0,0};
				int y_offset[]={0,0,0,1,-1};
				int offsetSize=sizeof(x_offset)/sizeof(int);
				if (x>=0&&x<m_width&&y>=0&&y<m_height){
					assert(labels_ind>=-1&&labels_ind<kseedsSize);
					{
						double l, a, b;
						double dist;
						double sita_n, m_n, L_n, X_n, Y_n;
						double Ksita_n, Km_n, KL_n, KX_n, KY_n;
						//////////////////////////////////////////////
						int i = y*m_width + x;
						////////////////////////////////////////////
						sita_n=dev_sita_n[i];
						m_n=dev_m_n[i];
						L_n=dev_L_n[i];
						X_n=dev_X_n[i];
						Y_n=dev_Y_n[i];
						int kseedareaXmin=max(0,x-STEP);
						int kseedareaYmin=max(0,y-STEP);
						int kseedareaXmax=min(m_width,x+STEP);
						int kseedareaYmax=min(m_height,y+STEP);

						for (int xi=0;xi<offsetSize;xi++){
							for (int yi=0;yi<offsetSize;yi++)
							{
								int nx=labelsX+xi;
								int ny=labelsY+yi;
								if (nx>=0&&nx<SeedWH&&ny>=0&&ny<SeedWH){
									int ni=nx+ny*SeedWH;
									{
										int KseedX=kseedsx[ni];
										int KseedY=kseedsy[ni];
										if ((KseedX>=kseedareaXmin)&&
											(KseedX<kseedareaXmax)&&
											(KseedY>=kseedareaYmin)&&
											(KseedY<kseedareaYmax)){
												Ksita_n=dev_sita_n[ni];
												Km_n=dev_m_n[ni];
												KL_n=dev_L_n[ni];
												KX_n=dev_X_n[ni];
												KY_n=dev_Y_n[ni];
												dist=CalculateNewDistance(alpha,betta,gama,fai,sita_n,m_n,L_n,X_n,Y_n,Ksita_n,Km_n,KL_n,KX_n,KY_n);
												if( dist < distvec[i] )
												{
													distvec[i] = dist;
													klabels[i]  = ni;
												}
										}
									}
								}

							}
						}
					}
				}
			}
#endif
		}
	}
#endif
	////////////////////////////////////////////////////////////////////////
#if 0
	{
		int x =blockIdx.x*blockDim.x+threadIdx.x;
		int y =blockIdx.y*blockDim.y+threadIdx.y;
		int SeedWH=sqrt((float)kseedsSize);
		int ind=x+y*m_width;
		int x_offset[]={0,1,-1,0,0};
		int y_offset[]={0,0,0,1,-1};
		int offsetSize=sizeof(x_offset)/sizeof(int);
		int labels_ind=klabels[ind];
		if (labels_ind==-1){
			int x_step=m_width/SeedWH;
			int y_step=m_height/SeedWH;
			int labelsX=x/x_step;
			int labelsY=y/y_step;
#if 1
			if (x>=0&&x<m_width&&y>=0&&y<m_height){
				assert(labels_ind>=-1&&labels_ind<kseedsSize);
				{
					double l, a, b;
					double dist;
					double sita_n, m_n, L_n, X_n, Y_n;
					double Ksita_n, Km_n, KL_n, KX_n, KY_n;
					//////////////////////////////////////////////
					int i = y*m_width + x;
					////////////////////////////////////////////
					sita_n=dev_sita_n[i];
					m_n=dev_m_n[i];
					L_n=dev_L_n[i];
					X_n=dev_X_n[i];
					Y_n=dev_Y_n[i];
					int kseedareaXmin=max(0,x-STEP);
					int kseedareaYmin=max(0,y-STEP);
					int kseedareaXmax=min(m_width,x+STEP);
					int kseedareaYmax=min(m_height,y+STEP);
					for (int xi=0;xi<offsetSize;xi++){
						for (int yi=0;yi<offsetSize;yi++)
						{
							int nx=labelsX+x_offset[xi];
							int ny=labelsY+y_offset[yi];
							if (nx>=0&&nx<SeedWH&&ny>=0&&ny<SeedWH){
								int ni=nx+ny*SeedWH;
								{
									int KseedX=kseedsx[ni];
									int KseedY=kseedsy[ni];
									if ((KseedX>=kseedareaXmin)&&
										(KseedX<kseedareaXmax)&&
										(KseedY>=kseedareaYmin)&&
										(KseedY<kseedareaYmax)){
											Ksita_n=dev_sita_n[ni];
											Km_n=dev_m_n[ni];
											KL_n=dev_L_n[ni];
											KX_n=dev_X_n[ni];
											KY_n=dev_Y_n[ni];
											dist=CalculateNewDistance(alpha,betta,gama,fai,sita_n,m_n,L_n,X_n,Y_n,Ksita_n,Km_n,KL_n,KX_n,KY_n);
											if( dist < distvec[i] )
											{
												distvec[i] = dist;
												klabels[i]  = ni;
											}
									}
								}
							}
						}
					}
				}
			}
#endif
		}else{
			int labelsX=labels_ind%SeedWH;
			int labelsY=labels_ind/SeedWH;
#if 1
			if (x>=0&&x<m_width&&y>=0&&y<m_height){
				assert(labels_ind>=-1&&labels_ind<kseedsSize);
				{
					double l, a, b;
					double dist;
					double sita_n, m_n, L_n, X_n, Y_n;
					double Ksita_n, Km_n, KL_n, KX_n, KY_n;
					//////////////////////////////////////////////
					int i = y*m_width + x;
					////////////////////////////////////////////
					sita_n=dev_sita_n[i];
					m_n=dev_m_n[i];
					L_n=dev_L_n[i];
					X_n=dev_X_n[i];
					Y_n=dev_Y_n[i];
					int kseedareaXmin=max(0,x-STEP);
					int kseedareaYmin=max(0,y-STEP);
					int kseedareaXmax=min(m_width,x+STEP);
					int kseedareaYmax=min(m_height,y+STEP);
					for (int xi=0;xi<offsetSize;xi++){
						for (int yi=0;yi<offsetSize;yi++)
						{
							int nx=labelsX+x_offset[xi];
							int ny=labelsY+y_offset[yi];
							if (nx>=0&&nx<SeedWH&&ny>=0&&ny<SeedWH){
								int ni=nx+ny*SeedWH;
								{
									int KseedX=kseedsx[ni];
									int KseedY=kseedsy[ni];
									if ((KseedX>=kseedareaXmin)&&
										(KseedX<kseedareaXmax)&&
										(KseedY>=kseedareaYmin)&&
										(KseedY<kseedareaYmax)){
											Ksita_n=dev_sita_n[ni];
											Km_n=dev_m_n[ni];
											KL_n=dev_L_n[ni];
											KX_n=dev_X_n[ni];
											KY_n=dev_Y_n[ni];
											dist=CalculateNewDistance(alpha,betta,gama,fai,sita_n,m_n,L_n,X_n,Y_n,Ksita_n,Km_n,KL_n,KX_n,KY_n);
											if( dist < distvec[i] )
											{
												distvec[i] = dist;
												klabels[i]  = ni;
											}
									}
								}
							}
						}
					}
				}
			}
#endif
		}
	}
#endif
}
/*------------------------------------------------------------------------------------------*/
/**
*
*
*
*/
/*------------------------------------------------------------------------------------------*/
__global__ void SetMatrix(double *matrix,int width,int height,double value_t)
{
	int i=blockIdx.x*blockDim.x+threadIdx.x;
	int j=blockIdx.y*blockDim.y+threadIdx.y;
	int ind=i+j*width;
	if(ind<width*height){
		matrix[ind]=value_t;
	}
}
/*------------------------------------------------------------------------------------------*/
/**
*
*
*
*/
/*------------------------------------------------------------------------------------------*/
__global__ void SetMatrix_Efficient(double *matrix,int width,int height,double value_t)
{
	int ind=blockIdx.x*blockDim.x+threadIdx.x;
	int offset=blockDim.x*gridDim.x;
	int size_t=width*height;
	/////////////////////////////////////////
	while(ind<size_t){
		matrix[ind]=value_t;
		ind+=offset;
	}
	/////////////////////////////////////////
}
/*------------------------------------------------------------------------------------------*/
/**
*
*
*
*/
/*------------------------------------------------------------------------------------------*/
__device__ double atomicAdd(double* address, double val)
{
	unsigned long long int* address_as_ull =
		(unsigned long long int*)address;
	unsigned long long int old = *address_as_ull, assumed;

	do {
		assumed = old;
		old = atomicCAS(address_as_ull, assumed,
			__double_as_longlong(val +
			__longlong_as_double(assumed)));

		// Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
	} while (assumed != old);

	return __longlong_as_double(old);
}
//Read more at: http://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#ixzz3HQ3voVZ1 
//Follow us: @GPUComputing on Twitter | NVIDIA on Facebook
/*------------------------------------------------------------------------------------------*/
/**
*              1*1            
*
*
*/
/*------------------------------------------------------------------------------------------*/
__global__ void StatisticsLABXY_Seed_collect_blocks(
	const int			kseedSize,
	const int			width,
	const int			height,
	double*				m_lvec,
	double*				m_avec,
	double*				m_bvec,
	int*				dev_klabels,
	double*				dev_sigmal,
	double*				dev_sigmaa,
	double*				dev_sigmab,
	double*				dev_sigmax,
	double*				dev_sigmay,
	double*				dev_clustersize)
{
	int m=blockIdx.x*blockDim.x+threadIdx.x;
	int n=blockIdx.y*blockDim.y+threadIdx.y;
	///////////////////////////////////////
	int m_dim=blockDim.x*gridDim.x;
	int n_dim=blockDim.y*gridDim.y;
	int width_step=width/m_dim;
	int height_step=height/n_dim;
	/////////////////////////////////////////
	int w_start=m*width_step;
	int h_start=n*height_step;
	int w_end=min((m+1)*width_step,width);
	int h_end=min((n+1)*height_step,height);
	///////////////////////////////////////////
	int labels_ind;
	int ind;
	for (int xi=w_start;xi<w_end;xi++){
		for (int yi=h_start;yi<h_end;yi++){
			ind=xi+yi*width;
			labels_ind=dev_klabels[ind];
			assert(labels_ind>=0&&labels_ind<kseedSize);
#if 1
					{
						atomicAdd(&dev_sigmal[labels_ind],m_lvec[ind]);
						atomicAdd(&dev_sigmaa[labels_ind],m_avec[ind]);
						atomicAdd(&dev_sigmab[labels_ind],m_bvec[ind]);
						atomicAdd(&dev_sigmax[labels_ind],xi);
						atomicAdd(&dev_sigmay[labels_ind],yi);
						atomicAdd(&dev_clustersize[labels_ind],1.0);
					}
#else
						{
							dev_sigmal[labels_ind]+=m_lvec[ind];
							dev_sigmaa[labels_ind]+=m_avec[ind];
							dev_sigmab[labels_ind]+=m_bvec[ind];
							dev_sigmax[labels_ind]+=xi;
							dev_sigmay[labels_ind]+=yi;
							dev_clustersize[labels_ind]+=1.0;
						}
#endif
		}
	}
}
/*------------------------------------------------------------------------------------------*/
/**
*              1*1            
*
*
*/
/*------------------------------------------------------------------------------------------*/
__global__ void StatisticsLABXY_Seed_collect_blocks_sheard(
	const int			kseedSize,
	const int			width,
	const int			height,
	double*				m_lvec,
	double*				m_avec,
	double*				m_bvec,
	int*				dev_klabels,
	double*				dev_sigmal,
	double*				dev_sigmaa,
	double*				dev_sigmab,
	double*				dev_sigmax,
	double*				dev_sigmay,
	double*				dev_clustersize)
{
	int m=blockIdx.x*blockDim.x+threadIdx.x;
	int n=blockIdx.y*blockDim.y+threadIdx.y;
	///////////////////////////////////////
	int m_dim=blockDim.x*gridDim.x;
	int n_dim=blockDim.y*gridDim.y;
	int width_step=width/m_dim;
	int height_step=height/n_dim;
	/////////////////////////////////////////
	int w_start=m*width_step;
	int h_start=n*height_step;
	int w_end=min((m+1)*width_step,width);
	int h_end=min((n+1)*height_step,height);
	///////////////////////////////////////////
	int labels_ind;
	int ind;
	for (int xi=w_start;xi<w_end;xi++){
		for (int yi=h_start;yi<h_end;yi++){
			ind=xi+yi*width;
			labels_ind=dev_klabels[ind];
			assert(labels_ind>=0&&labels_ind<kseedSize);
#if 1
			{
				atomicAdd(&dev_sigmal[labels_ind],m_lvec[ind]);
				atomicAdd(&dev_sigmaa[labels_ind],m_avec[ind]);
				atomicAdd(&dev_sigmab[labels_ind],m_bvec[ind]);
				atomicAdd(&dev_sigmax[labels_ind],xi);
				atomicAdd(&dev_sigmay[labels_ind],yi);
				atomicAdd(&dev_clustersize[labels_ind],1.0);
			}
#else
			{
				dev_sigmal[labels_ind]+=m_lvec[ind];
				dev_sigmaa[labels_ind]+=m_avec[ind];
				dev_sigmab[labels_ind]+=m_bvec[ind];
				dev_sigmax[labels_ind]+=xi;
				dev_sigmay[labels_ind]+=yi;
				dev_clustersize[labels_ind]+=1.0;
			}
#endif
		}
	}
}
/*------------------------------------------------------------------------------------------*/
/**
*
*
*
*/
/*------------------------------------------------------------------------------------------*/
__global__ void StatisticsLABXY_Seed_collect(
	unsigned int*		mutex,
	const int			kseedSize,
	const int			width,
	const int			height,
	double*				m_lvec,
	double*				m_avec,
	double*				m_bvec,
	int*				dev_klabels,
	double*				dev_sigmal,
	double*				dev_sigmaa,
	double*				dev_sigmab,
	double*				dev_sigmax,
	double*				dev_sigmay,
	double*				dev_clustersize)
{
	int c=blockIdx.x*blockDim.x+threadIdx.x;
	int r=blockIdx.y*blockDim.y+threadIdx.y;
	int ind=c+r*width;
	int labels_ind;
	if (ind>=0&&ind<width*height){
		labels_ind=dev_klabels[ind];
	}else{
		return;
	}

	assert(labels_ind>=0&&labels_ind<kseedSize);
	//	bool finished = false; 
	if (labels_ind>=0&&labels_ind<kseedSize)
	{
		if (ind<width*height){

			//	while(!finished)
			{
				//	if( atomicExch(mutex, 1) == 0)
				{
#if 1
					{
						atomicAdd(&dev_sigmal[labels_ind],m_lvec[ind]);
						atomicAdd(&dev_sigmaa[labels_ind],m_avec[ind]);
						atomicAdd(&dev_sigmab[labels_ind],m_bvec[ind]);
						atomicAdd(&dev_sigmax[labels_ind],c);
						atomicAdd(&dev_sigmay[labels_ind],r);
						atomicAdd(&dev_clustersize[labels_ind],1.0);
					}
#else
					{
						dev_sigmal[labels_ind]+=m_lvec[ind];
						dev_sigmaa[labels_ind]+=m_avec[ind];
						dev_sigmab[labels_ind]+=m_bvec[ind];
						dev_sigmax[labels_ind]+=c;
						dev_sigmay[labels_ind]+=r;
						dev_clustersize[labels_ind]+=1.0;
					}
#endif

					//	finished = true;
					//	atomicExch(mutex, 0);
				}
			}

		}
	}
	/*****************************************/
	//while(!finished)
	//{
	//		if( atomicExch(mutex, 1) == 0){
	//			
	//			
	//			finished = true;
	//			atomicExch(mutex, 0);
	//		}

	//}

	/*****************************************/

}
/*------------------------------------------------------------------------------------------*/
/**
*
*�ռ�ȫͼ�����ӣ�Ĭ����K^2��ƽ����������
*
*/
/*------------------------------------------------------------------------------------------*/
__global__ void StatisticsLABXY_Seed_collect_new2014_11_3(
	unsigned int*		mutex,
	const int			kseedSize,
	const int			width,
	const int			height,
	double*				m_lvec,
	double*				m_avec,
	double*				m_bvec,
	int*				dev_klabels,
	double*				dev_sigmal,
	double*				dev_sigmaa,
	double*				dev_sigmab,
	double*				dev_sigmax,
	double*				dev_sigmay,
	double*				dev_clustersize)
{
	__shared__ double temp[4096];
	assert(kseedSize<=4096);
	{
		int ind=threadIdx.x+blockIdx.x*blockDim.x;
		int offset=blockDim.x*gridDim.x;
		temp[threadIdx.x]=temp[threadIdx.x+1024]=temp[threadIdx.x+2048]=temp[threadIdx.x+3072]=0;
			__syncthreads();
		while(ind<width*height){
			int labels_ind=dev_klabels[ind];
			assert(labels_ind>=0&&labels_ind<kseedSize);
			atomicAdd(&temp[labels_ind],m_lvec[ind]);
			ind+=offset;
		}
			__syncthreads();
		atomicAdd(&dev_sigmal[threadIdx.x],temp[threadIdx.x]);
		atomicAdd(&dev_sigmal[threadIdx.x+1024],temp[threadIdx.x+1024]);
		atomicAdd(&dev_sigmal[threadIdx.x+2048],temp[threadIdx.x+2048]);
		atomicAdd(&dev_sigmal[threadIdx.x+3072],temp[threadIdx.x+3072]);
	}
#if 0
{
		int ind=threadIdx.x+blockIdx.x*blockDim.x;
		int offset=blockDim.x*gridDim.x;
		temp[threadIdx.x]=temp[threadIdx.x+1024]=temp[threadIdx.x+2048]=temp[threadIdx.x+3072]=0;
			__syncthreads();
		while(ind<width*height){
			int labels_ind=dev_klabels[ind];
			assert(labels_ind>=0&&labels_ind<kseedSize);
			atomicAdd(&temp[labels_ind],m_avec[ind]);
			ind+=offset;
		}
			__syncthreads();
		atomicAdd(&dev_sigmaa[threadIdx.x],temp[threadIdx.x]);
		atomicAdd(&dev_sigmaa[threadIdx.x+1024],temp[threadIdx.x+1024]);
		atomicAdd(&dev_sigmaa[threadIdx.x+2048],temp[threadIdx.x+2048]);
		atomicAdd(&dev_sigmaa[threadIdx.x+3072],temp[threadIdx.x+3072]);
	}
	{
		int ind=threadIdx.x+blockIdx.x*blockDim.x;
		int offset=blockDim.x*gridDim.x;
		temp[threadIdx.x]=temp[threadIdx.x+1024]=temp[threadIdx.x+2048]=temp[threadIdx.x+3072]=0;
			__syncthreads();
		while(ind<width*height){
			int labels_ind=dev_klabels[ind];
			assert(labels_ind>=0&&labels_ind<kseedSize);
				atomicAdd(&temp[labels_ind],m_bvec[ind]);
			ind+=offset;
		}
			__syncthreads();
		atomicAdd(&dev_sigmab[threadIdx.x],temp[threadIdx.x]);
		atomicAdd(&dev_sigmab[threadIdx.x+1024],temp[threadIdx.x+1024]);
		atomicAdd(&dev_sigmab[threadIdx.x+2048],temp[threadIdx.x+2048]);
		atomicAdd(&dev_sigmab[threadIdx.x+3072],temp[threadIdx.x+3072]);
	}
	{
		int ind=threadIdx.x+blockIdx.x*blockDim.x;
		int offset=blockDim.x*gridDim.x;
		temp[threadIdx.x]=temp[threadIdx.x+1024]=temp[threadIdx.x+2048]=temp[threadIdx.x+3072]=0;
			__syncthreads();
		while(ind<width*height){
			int labels_ind=dev_klabels[ind];
			assert(labels_ind>=0&&labels_ind<kseedSize);
				atomicAdd(&temp[labels_ind],int(ind%width));
			ind+=offset;
		}
			__syncthreads();
		atomicAdd(&dev_sigmax[threadIdx.x],temp[threadIdx.x]);
		atomicAdd(&dev_sigmax[threadIdx.x+1024],temp[threadIdx.x+1024]);
		atomicAdd(&dev_sigmax[threadIdx.x+2048],temp[threadIdx.x+2048]);
		atomicAdd(&dev_sigmax[threadIdx.x+3072],temp[threadIdx.x+3072]);
	}
	{
		int ind=threadIdx.x+blockIdx.x*blockDim.x;
		int offset=blockDim.x*gridDim.x;
		temp[threadIdx.x]=temp[threadIdx.x+1024]=temp[threadIdx.x+2048]=temp[threadIdx.x+3072]=0;
			__syncthreads();
		while(ind<width*height){
			int labels_ind=dev_klabels[ind];
			assert(labels_ind>=0&&labels_ind<kseedSize);
				atomicAdd(&temp[labels_ind],int(ind/width));
			ind+=offset;
		}
			__syncthreads();
		atomicAdd(&dev_sigmay[threadIdx.x],temp[threadIdx.x]);
		atomicAdd(&dev_sigmay[threadIdx.x+1024],temp[threadIdx.x+1024]);
		atomicAdd(&dev_sigmay[threadIdx.x+2048],temp[threadIdx.x+2048]);
		atomicAdd(&dev_sigmay[threadIdx.x+3072],temp[threadIdx.x+3072]);
	}
	{
		int ind=threadIdx.x+blockIdx.x*blockDim.x;
		int offset=blockDim.x*gridDim.x;
		temp[threadIdx.x]=temp[threadIdx.x+1024]=temp[threadIdx.x+2048]=temp[threadIdx.x+3072]=0;
			__syncthreads();
		while(ind<width*height){
			int labels_ind=dev_klabels[ind];
			assert(labels_ind>=0&&labels_ind<kseedSize);
				atomicAdd(&temp[labels_ind],1);
			ind+=offset;
		}
			__syncthreads();
		atomicAdd(&dev_clustersize[threadIdx.x],temp[threadIdx.x]);
		atomicAdd(&dev_clustersize[threadIdx.x+1024],temp[threadIdx.x+1024]);
		atomicAdd(&dev_clustersize[threadIdx.x+2048],temp[threadIdx.x+2048]);
		atomicAdd(&dev_clustersize[threadIdx.x+3072],temp[threadIdx.x+3072]);
	}
#endif	
}
/*------------------------------------------------------------------------------------------*/
/**
*
*�ռ�ȫͼ�����ӣ�Ĭ����K^2��ƽ����������
*
*@param			kseedsSize		��������
*@param			width		ͼ����
*@param			height		ͼ��߶�
*@param	[in]	m_lvec		ͼ��L��������
*@param	[in]	m_avec		ͼ��A��������
*@param	[in]	m_bvec		ͼ��B��������
*@param	[in]	dev_klabels		ͼ��궨����
*@param [in]	dev_img_mask		δʹ��
*@param	[out]	dev_sigmal		��L����ͳ�����
*@param	[out]	dev_sigmaa		��A����ͳ�����
*@param	[out]	dev_sigmab		��B����ͳ�����
*@param	[out]	dev_sigmax		��X����ͳ�����
*@param	[out]	dev_sigmay		��Y����ͳ�����
*@param	[out]	dev_clustersize	 δʹ�� 	
*
*/
/*------------------------------------------------------------------------------------------*/
 void StatisticsLABXY_Seed_collect_CPU(
	const int			kseedsSize,
	const int			width,
	const int			height,
	double*				m_lvec,
	double*				m_avec,
	double*				m_bvec,
	int*				dev_klabels,
	unsigned int*       dev_img_mask,
	double*				dev_sigmal,
	double*				dev_sigmaa,
	double*				dev_sigmab,
	double*				dev_sigmax,
	double*				dev_sigmay,
	double*				dev_clustersize)
{
	hipError_t cudaStatus;
	unsigned int*       host_img_mask=nullptr;
	int*				host_klabels=new int[width*height];
	double*				host_sigmal=new double[kseedsSize];
	double*				host_sigmaa=new double[kseedsSize];
	double*				host_sigmab=new double[kseedsSize];
	double*				host_sigmax=new double[kseedsSize];
	double*				host_sigmay=new double[kseedsSize];
	double*				host_clustersize=new double[kseedsSize];

#if 1
/////////////////////////////////////////////////////////////////////////////////////////////////////////
	if (dev_img_mask==nullptr){
		host_img_mask=nullptr;
	}else{
		 host_img_mask=new unsigned int[width*height];
		 cudaStatus = hipMemcpy( host_img_mask,dev_img_mask, width*height * sizeof(unsigned int), hipMemcpyDeviceToHost);
	}
	 cudaStatus = hipMemcpy(host_klabels,dev_klabels, width*height * sizeof(int), hipMemcpyDeviceToHost);
	memset(host_sigmal,0,kseedsSize * sizeof(double));
	memset(host_sigmaa,0,kseedsSize * sizeof(double));
	memset(host_sigmab,0,kseedsSize * sizeof(double));
	memset(host_sigmax,0,kseedsSize * sizeof(double));
	memset(host_sigmay,0,kseedsSize * sizeof(double));	
	memset(host_clustersize,0,kseedsSize*sizeof(double));
	/////////////////////////////////////////////////////////////////////////////////////////////////////////
#endif
	{
		int ind(0);
		for(register int r = 0; r < height; r++ )
		{
			for(register  int c = 0; c < width; c++ )
			{
				if (host_img_mask==nullptr){
					host_sigmal[host_klabels[ind]] += m_lvec[ind];
					host_sigmaa[host_klabels[ind]] += m_avec[ind];
					host_sigmab[host_klabels[ind]] += m_bvec[ind];
					host_sigmax[host_klabels[ind]] += c;
					host_sigmay[host_klabels[ind]] += r;
					host_clustersize[host_klabels[ind]] += 1.0;
				}
				if (host_img_mask!=nullptr&&host_img_mask[ind]==0x000000000){
					host_sigmal[host_klabels[ind]] += m_lvec[ind];
					host_sigmaa[host_klabels[ind]] += m_avec[ind];
					host_sigmab[host_klabels[ind]] += m_bvec[ind];
					host_sigmax[host_klabels[ind]] += c;
					host_sigmay[host_klabels[ind]] += r;
					host_clustersize[host_klabels[ind]] += 1.0;
				}				
				//------------------------------------
				//edgesum[klabels[ind]] += edgemag[ind];
				//------------------------------------
				
				ind++;
			}
		}
#if 0
				double x_max=0;
				double y_max=0;
				double *inv=new double[kseedsSize];
				{
					for( int k = 0; k < kseedsSize; k++ )
					{
						if(host_clustersize[k] <= 0 ) host_clustersize[k] = 1;
						inv[k] = 1.0/host_clustersize[k];//computing inverse now to multiply, than divide later
						////////////////////////////////////
						host_sigmal[k] = host_sigmal[k]*inv[k];
						host_sigmaa[k] = host_sigmaa[k]*inv[k];
						host_sigmab[k] = host_sigmab[k]*inv[k];
						host_sigmax[k] = host_sigmax[k]*inv[k];
						host_sigmay[k] = host_sigmay[k]*inv[k];
						if (host_sigmax[k]>x_max){
							x_max=host_sigmax[k];
						}
						if (host_sigmay[k]>y_max){
							y_max=host_sigmay[k];
						}
						//------------------------------------
						//edgesum[k] *= inv[k];
						//------------------------------------
					}
				}
				delete []inv;
#endif
	}
#if 1
	/////////////////////////////////////////////////////////////////////////////////////////////////////////
	if (dev_img_mask==nullptr){
		host_img_mask=nullptr;
	}else{
		cudaStatus = hipMemcpy( dev_img_mask,host_img_mask, width*height * sizeof(int), hipMemcpyHostToDevice);
	}
	
	
	cudaStatus = hipMemcpy(dev_sigmal, host_sigmal, kseedsSize * sizeof(double), hipMemcpyHostToDevice);
	
	cudaStatus = hipMemcpy(dev_sigmaa, host_sigmaa, kseedsSize * sizeof(double), hipMemcpyHostToDevice);
	
	cudaStatus = hipMemcpy(dev_sigmab, host_sigmab, kseedsSize * sizeof(double), hipMemcpyHostToDevice);
	
	cudaStatus = hipMemcpy(dev_sigmax, host_sigmax, kseedsSize * sizeof(double), hipMemcpyHostToDevice);
	
	cudaStatus = hipMemcpy(dev_sigmay, host_sigmay, kseedsSize * sizeof(double), hipMemcpyHostToDevice);
	
	cudaStatus = hipMemcpy(dev_clustersize, host_clustersize, kseedsSize * sizeof(double), hipMemcpyHostToDevice);

	/////////////////////////////////////////////////////////////////////////////////////////////////////////
#endif
	delete [] host_img_mask;
	delete [] host_klabels;
	delete [] host_sigmal;
	delete [] host_sigmaa;
	delete [] host_sigmab;
	delete [] host_sigmax;
	delete [] host_sigmay;
	delete [] host_clustersize;
 ;
}
/*------------------------------------------------------------------------------------------*/
/**
*
*�ռ�ȫͼ�����ӣ�Ĭ����K^2��ƽ����������
*
*@param			kseedsSize		��������
*@param			width		ͼ����
*@param			height		ͼ��߶�
*@param	[in]	m_lvec		ͼ��L��������
*@param	[in]	m_avec		ͼ��A��������
*@param	[in]	m_bvec		ͼ��B��������
*@param	[in]	dev_klabels		ͼ��궨����
*@param [in]	dev_img_mask		δʹ��
*@param	[out]	dev_sigmal		��L����ͳ�����
*@param	[out]	dev_sigmaa		��A����ͳ�����
*@param	[out]	dev_sigmab		��B����ͳ�����
*@param	[out]	dev_sigmax		��X����ͳ�����
*@param	[out]	dev_sigmay		��Y����ͳ�����
*@param	[out]	dev_clustersize	 δʹ�� 	
*
*/
/*------------------------------------------------------------------------------------------*/
 void StatisticsThetaMLXY_Seed_collect_CPU(
	const int			kseedsSize,
	const int			width,
	const int			height,
	double*       host_sita_n,
	double*       host_m_n,
	double*       host_L_n,
	double*       host_X_n,
	double*       host_Y_n,	
	int*				dev_klabels,
    double*	      dev_kseed_sita_n,
	double*	      dev_kseed_m_n,
	double*	      dev_kseed_L_n,
	double*       dev_kseed_X_n,
	double*	      dev_kseed_Y_n)
{
	hipError_t cudaStatus;
	double*	      host_kseed_sita_n=new double[kseedsSize];
	double*	      host_kseed_m_n=new double[kseedsSize];
	double*	      host_kseed_L_n=new double[kseedsSize];
	double*       host_kseed_X_n=new double[kseedsSize];
	double*	      host_kseed_Y_n=new double[kseedsSize];
	int*	      host_klabels=new int[width*height];
	unsigned int*       dev_img_mask;
	unsigned int*       host_img_mask=new unsigned[width*height];
 
   cudaStatus=hipMalloc((void**)&dev_img_mask,sizeof(unsigned int)*width*height);  
   InitMaskTemplate(dev_img_mask,dev_klabels,width,height,4);

   cudaStatus = hipMemcpy(host_kseed_sita_n, dev_kseed_sita_n, kseedsSize * sizeof(double), hipMemcpyDeviceToHost);
   cudaStatus = hipMemcpy(host_kseed_m_n, dev_kseed_m_n, kseedsSize * sizeof(double), hipMemcpyDeviceToHost);
   cudaStatus = hipMemcpy(host_kseed_L_n, dev_kseed_L_n, kseedsSize * sizeof(double), hipMemcpyDeviceToHost);
   cudaStatus = hipMemcpy(host_kseed_X_n, dev_kseed_X_n, kseedsSize * sizeof(double), hipMemcpyDeviceToHost);
   cudaStatus = hipMemcpy(host_kseed_Y_n, dev_kseed_Y_n, kseedsSize * sizeof(double), hipMemcpyDeviceToHost);
   cudaStatus = hipMemcpy(host_klabels, dev_klabels, kseedsSize * sizeof(double), hipMemcpyDeviceToHost);
   cudaStatus = hipMemcpy(host_img_mask,dev_img_mask, kseedsSize * sizeof(double), hipMemcpyDeviceToHost);
  
	StatisticsThetaMLXY_Seed_collect_CPU_ALL(
					kseedsSize,
					width,
					height,
					host_sita_n,
					host_m_n,
					host_L_n,
					host_X_n,
					host_Y_n,	
					host_klabels,
					host_img_mask,
					host_kseed_sita_n,
					host_kseed_m_n,
					host_kseed_L_n,
					host_kseed_X_n,
					host_kseed_Y_n);

    delete[]	  host_kseed_sita_n;
	delete[]      host_kseed_m_n;
	delete[]      host_kseed_L_n;
	delete[]      host_kseed_X_n;
	delete[]      host_kseed_Y_n;
	delete[]      host_klabels;
	hipFree(dev_img_mask);
	delete[]      host_img_mask;

}
/*------------------------------------------------------------------------------------------*/
/**
*
*�ռ�ȫͼ�����ӣ�Ĭ����K^2��ƽ����������
*
*@param			kseedsSize		��������
*@param			width		ͼ����
*@param			height		ͼ��߶�
*@param	[in]	m_lvec		ͼ��L��������
*@param	[in]	m_avec		ͼ��A��������
*@param	[in]	m_bvec		ͼ��B��������
*@param	[in]	dev_klabels		ͼ��궨����
*@param [in]	dev_img_mask		δʹ��
*@param	[out]	dev_sigmal		��L����ͳ�����
*@param	[out]	dev_sigmaa		��A����ͳ�����
*@param	[out]	dev_sigmab		��B����ͳ�����
*@param	[out]	dev_sigmax		��X����ͳ�����
*@param	[out]	dev_sigmay		��Y����ͳ�����
*@param	[out]	dev_clustersize	 δʹ�� 	
*
*/
/*------------------------------------------------------------------------------------------*/
void StatisticsThetaMLXY_Seed_collect_CPU_ALL(
	const int			kseedsSize,
	const int			width,
	const int			height,
	double*       host_sita_n,
	double*       host_m_n,
	double*       host_L_n,
	double*       host_X_n,
	double*       host_Y_n,	
	int*				host_klabels,
	unsigned int*       host_img_mask,
    double*	      host_kseed_sita_n,
	double*	      host_kseed_m_n,
	double*	      host_kseed_L_n,
	double*       host_kseed_X_n,
	double*	      host_kseed_Y_n)
{
      double*			  host_sigmaSita=new double[kseedsSize];
	  double*			  host_sigmaM=new double[kseedsSize];
	  double*			  host_sigmaL=new double[kseedsSize];
	  double*			  host_sigmaX=new double[kseedsSize];
	  double*		      host_sigmaY=new double[kseedsSize];
	  double*			  host_clustersize=new double[kseedsSize];
	memset(host_sigmaSita,0,kseedsSize * sizeof(double));
	memset(host_sigmaM,0,kseedsSize * sizeof(double));
	memset(host_sigmaL,0,kseedsSize * sizeof(double));
	memset(host_sigmaX,0,kseedsSize * sizeof(double));
	memset(host_sigmaY,0,kseedsSize * sizeof(double));	
	memset(host_clustersize,0,kseedsSize*sizeof(double));
	{
		int ind(0);
		for(register int r = 0; r < height; r++ )
		{
			for(register  int c = 0; c < width; c++ )
			{
				if (host_img_mask==nullptr){
					host_sigmaSita[host_klabels[ind]] +=host_sita_n[ind];
					host_sigmaM[host_klabels[ind]] += host_m_n[ind];
					host_sigmaL[host_klabels[ind]] += host_L_n[ind];
					host_sigmaX[host_klabels[ind]] += host_X_n[ind];
					host_sigmaY[host_klabels[ind]] += host_Y_n[ind];
					host_clustersize[host_klabels[ind]] += 1.0;
				}
				if (host_img_mask!=nullptr&&host_img_mask[ind]==0x000000000){
					host_sigmaSita[host_klabels[ind]] +=host_sita_n[ind];
					host_sigmaM[host_klabels[ind]] += host_m_n[ind];
					host_sigmaL[host_klabels[ind]] += host_L_n[ind];
					host_sigmaX[host_klabels[ind]] += host_X_n[ind];
					host_sigmaY[host_klabels[ind]] += host_Y_n[ind];
					host_clustersize[host_klabels[ind]] += 1.0;
				}				
				//------------------------------------
				//edgesum[klabels[ind]] += edgemag[ind];
				//------------------------------------
				
				ind++;
			}
		}
		///////////////////////////////////////////////
		for(int spi=0;spi<kseedsSize;spi++){
			   host_kseed_sita_n[spi]=host_sigmaSita[spi]/host_clustersize[spi];
	           host_kseed_m_n[spi]=host_sigmaM[spi]/host_clustersize[spi];
	           host_kseed_L_n[spi]=host_sigmaL[spi]/host_clustersize[spi];
	           host_kseed_X_n[spi]=host_sigmaX[spi]/host_clustersize[spi];
	           host_kseed_Y_n[spi]=host_sigmaY[spi]/host_clustersize[spi];
		}
	}
	delete []  host_sigmaSita;
	delete []  host_sigmaM;
	delete []  host_sigmaL;
	delete []  host_sigmaX;
	delete []  host_sigmaY;
	delete []  host_clustersize;
}
/*------------------------------------------------------------------------------------------*/
/**
*
*�ռ�ȫͼ�����ӣ�Ĭ����K^2��ƽ����������
*
*@param			kseedsSize		��������
*@param			width		ͼ����
*@param			height		ͼ��߶�
*@param	[in]	m_lvec		ͼ��L��������
*@param	[in]	m_avec		ͼ��A��������
*@param	[in]	m_bvec		ͼ��B��������
*@param	[in]	dev_klabels		ͼ��궨����
*@param [in]	dev_img_mask		δʹ��
*@param	[out]	dev_sigmal		��L����ͳ�����
*@param	[out]	dev_sigmaa		��A����ͳ�����
*@param	[out]	dev_sigmab		��B����ͳ�����
*@param	[out]	dev_sigmax		��X����ͳ�����
*@param	[out]	dev_sigmay		��Y����ͳ�����
*@param	[out]	dev_clustersize	 δʹ�� 	
*
*/
/*------------------------------------------------------------------------------------------*/
void StatisticsThetaMLXY_Seed_collect_CPU_Incompletion(
	const int			kseedsSize,
	const int			width,
	const int			height,
	double*				m_lvec,
	double*				m_avec,
	double*				m_bvec,
	int*				dev_klabels,
	unsigned int*       dev_img_mask,
	double*	           dev_kseed_sita_n,
	double*	           dev_kseed_m_n,
	double*	           dev_kseed_L_n,
	double*            dev_kseed_X_n,
	double*	           dev_kseed_Y_n)
{
	
	hipError_t cudaStatus;
	unsigned int*       host_img_mask=nullptr;
	int*				host_klabels=new int[width*height];
	double*				host_sigmal=new double[kseedsSize];
	double*				host_sigmaa=new double[kseedsSize];
	double*				host_sigmab=new double[kseedsSize];
	double*				host_sigmax=new double[kseedsSize];
	double*				host_sigmay=new double[kseedsSize];
	double*				host_clustersize=new double[kseedsSize];

#if 1
/////////////////////////////////////////////////////////////////////////////////////////////////////////
	if (dev_img_mask==nullptr){
		host_img_mask=nullptr;
	}else{
		 host_img_mask=new unsigned int[width*height];
		 cudaStatus = hipMemcpy( host_img_mask,dev_img_mask, width*height * sizeof(unsigned int), hipMemcpyDeviceToHost);
	}
	 cudaStatus = hipMemcpy(host_klabels,dev_klabels, width*height * sizeof(int), hipMemcpyDeviceToHost);
	memset(host_sigmal,0,kseedsSize * sizeof(double));
	memset(host_sigmaa,0,kseedsSize * sizeof(double));
	memset(host_sigmab,0,kseedsSize * sizeof(double));
	memset(host_sigmax,0,kseedsSize * sizeof(double));
	memset(host_sigmay,0,kseedsSize * sizeof(double));	
	memset(host_clustersize,0,kseedsSize*sizeof(double));
	/////////////////////////////////////////////////////////////////////////////////////////////////////////
#endif
	{
		int ind(0);
		for(register int r = 0; r < height; r++ )
		{
			for(register  int c = 0; c < width; c++ )
			{
				if (host_img_mask==nullptr){
					host_sigmal[host_klabels[ind]] += m_lvec[ind];
					host_sigmaa[host_klabels[ind]] += m_avec[ind];
					host_sigmab[host_klabels[ind]] += m_bvec[ind];
					host_sigmax[host_klabels[ind]] += c;
					host_sigmay[host_klabels[ind]] += r;
					host_clustersize[host_klabels[ind]] += 1.0;
				}
				if (host_img_mask!=nullptr&&host_img_mask[ind]==0x000000000){
					host_sigmal[host_klabels[ind]] += m_lvec[ind];
					host_sigmaa[host_klabels[ind]] += m_avec[ind];
					host_sigmab[host_klabels[ind]] += m_bvec[ind];
					host_sigmax[host_klabels[ind]] += c;
					host_sigmay[host_klabels[ind]] += r;
					host_clustersize[host_klabels[ind]] += 1.0;
				}				
				//------------------------------------
				//edgesum[klabels[ind]] += edgemag[ind];
				//------------------------------------
				
				ind++;
			}
		}
#if 1
				
				double *inv=new double[kseedsSize];
				{
					for( int k = 0; k < kseedsSize; k++ )
					{
						if(host_clustersize[k] <= 0 ) host_clustersize[k] = 1;
						inv[k] = 1.0/host_clustersize[k];//computing inverse now to multiply, than divide later
						////////////////////////////////////
						host_sigmal[k] = host_sigmal[k]*inv[k];
						host_sigmaa[k] = host_sigmaa[k]*inv[k];
						host_sigmab[k] = host_sigmab[k]*inv[k];
						host_sigmax[k] = host_sigmax[k]*inv[k];
						host_sigmay[k] = host_sigmay[k]*inv[k];
						//------------------------------------
						//edgesum[k] *= inv[k];
						//------------------------------------
							ConvertLab2oml(
							width,
							height,
							host_sigmal[k],
							host_sigmaa[k],
							host_sigmab[k],
							host_sigmax[k],
							host_sigmay[k],
							&host_sigmal[k],
							&host_sigmaa[k],
							&host_sigmab[k],
							&host_sigmax[k],
							&host_sigmay[k]);
					}
				}
				delete []inv;
#endif
	}
#if 1
	/////////////////////////////////////////////////////////////////////////////////////////////////////////
	if (dev_img_mask==nullptr){
		host_img_mask=nullptr;
	}else{
		cudaStatus = hipMemcpy( dev_img_mask,host_img_mask, width*height * sizeof(int), hipMemcpyHostToDevice);
	}
	
	cudaStatus = hipMemcpy(dev_kseed_sita_n, host_sigmal, kseedsSize * sizeof(double), hipMemcpyHostToDevice);	
	cudaStatus = hipMemcpy(dev_kseed_m_n, host_sigmaa, kseedsSize * sizeof(double), hipMemcpyHostToDevice);	
	cudaStatus = hipMemcpy( dev_kseed_L_n, host_sigmab, kseedsSize * sizeof(double), hipMemcpyHostToDevice);	
	cudaStatus = hipMemcpy(dev_kseed_X_n, host_sigmax, kseedsSize * sizeof(double), hipMemcpyHostToDevice);	
	cudaStatus = hipMemcpy(dev_kseed_Y_n, host_sigmay, kseedsSize * sizeof(double), hipMemcpyHostToDevice);
	

	/////////////////////////////////////////////////////////////////////////////////////////////////////////
#endif
	delete [] host_img_mask;
	delete [] host_klabels;
	delete [] host_sigmal;
	delete [] host_sigmaa;
	delete [] host_sigmab;
	delete [] host_sigmax;
	delete [] host_sigmay;
	delete [] host_clustersize;
 ;
}
/*------------------------------------------------------------------------------------------*/
/**
*
*
*
*/
/*------------------------------------------------------------------------------------------*/
__global__ void StatisticsLABXY_Seed_collect_L(
	const int			kseedSize,
	const int			width,
	const int			height,
	double*				m_lvec,
	double*				m_avec,
	double*				m_bvec,
	int*				dev_klabels,
	double*				dev_sigmal,
	double*				dev_sigmaa,
	double*				dev_sigmab,
	double*				dev_sigmax,
	double*				dev_sigmay,
	double*				dev_clustersize)
{
	__shared__ double temp[4096];
	assert(kseedSize<=4096);
	{
		int ind=threadIdx.x+blockIdx.x*blockDim.x;
		int offset=blockDim.x*gridDim.x;
		temp[threadIdx.x]=temp[threadIdx.x+1024]=temp[threadIdx.x+2048]=temp[threadIdx.x+3072]=0;
		__syncthreads();
		while(ind<width*height){
			int labels_ind=dev_klabels[ind];
			assert(labels_ind>=0&&labels_ind<kseedSize);
			atomicAdd(&temp[labels_ind],m_lvec[ind]);
			ind+=offset;
		}
		__syncthreads();
		atomicAdd(&dev_sigmal[threadIdx.x],temp[threadIdx.x]);
		atomicAdd(&dev_sigmal[threadIdx.x+1024],temp[threadIdx.x+1024]);
		atomicAdd(&dev_sigmal[threadIdx.x+2048],temp[threadIdx.x+2048]);
		atomicAdd(&dev_sigmal[threadIdx.x+3072],temp[threadIdx.x+3072]);
	}
}
/*------------------------------------------------------------------------------------------*/
/**
*
*�ռ�ȫͼ�����ӣ�Ĭ����K^2��ƽ����������
*
*/
/*------------------------------------------------------------------------------------------*/
__global__ void StatisticsLABXY_Seed_collect_gpu_search(
	const int			kseedsSize,
	const int			width,
	const int			height,
	double*				kseedsx,
	double*				kseedsy,
	double*				dev_m_lvec,
	double*				dev_m_avec,
	double*				dev_m_bvec,
	int*				dev_klabels,
	double*				dev_sigmal,
	double*				dev_sigmaa,
	double*				dev_sigmab,
	double*				dev_sigmax,
	double*				dev_sigmay,
	double*				dev_clustersize)
{
	int SeedIdx=threadIdx.x+blockIdx.x*blockDim.x;
	int offset=blockDim.x*gridDim.x;

	while(SeedIdx<kseedsSize){
		////////////////////////////////////////
		int kseedWH=sqrt((float)kseedsSize);
		int x_step=width/(kseedWH);
		int y_step=height/(kseedWH);
		/////////////////////////////////////////
		int x_min=max((int)(kseedsx[SeedIdx]-x_step),(int)0);
		int x_max=min((int)(kseedsx[SeedIdx]+x_step),(int)width);
		int y_min=max((int)(kseedsy[SeedIdx]-y_step),(int)0);
		int y_max=min((int)(kseedsy[SeedIdx]+y_step),(int)height);
		/////////////////////////////////////////

		for (register int xi=x_min;xi<x_max;xi++){
			for (register int yi=y_min;yi<y_max;yi++){
				int ImgIdx=xi+yi*width;
				int labels_t=dev_klabels[ImgIdx];
#if 1
				if (labels_t==SeedIdx){
					dev_sigmal[SeedIdx] += dev_m_lvec[ImgIdx];
					dev_sigmaa[SeedIdx] += dev_m_avec[ImgIdx];
					dev_sigmab[SeedIdx] += dev_m_bvec[ImgIdx];
					dev_sigmax[SeedIdx] += xi;
					dev_sigmay[SeedIdx] += yi;
					dev_clustersize[SeedIdx] += 1.0;
				}
#endif
			}
		}

		
		/////////////////////////////////////////
			SeedIdx+=offset;
	}
	
	
}
/*------------------------------------------------------------------------------------------*/
/**
*
*�ռ�ȫͼ�����ӣ����㿪ʼȫͼ����
*
*/
/*------------------------------------------------------------------------------------------*/
__global__ void StatisticsLABXY_Seed_collect_gpu_search_AllImg(
	const int			kseedsSize,
	const int			width,
	const int			height,
	double*				dev_m_lvec,
	double*				dev_m_avec,
	double*				dev_m_bvec,
	int*				dev_klabels,
	double*				dev_sigmal,
	double*				dev_sigmaa,
	double*				dev_sigmab,
	double*				dev_sigmax,
	double*				dev_sigmay,
	double*				dev_clustersize)
{
	int SeedIdx=threadIdx.x+blockIdx.x*blockDim.x;
	int offset=blockDim.x*gridDim.x;
	while(SeedIdx<kseedsSize){
		/*--------------------------------------*/
		int kseedWH=sqrt((float)kseedsSize);
		int x_step=width/(kseedWH);
		int y_step=height/(kseedWH);
		/*--------------------------------------*/
		int x_min=0;
		int x_max=width;
		int y_min=0;
		int y_max=height;
		/*--------------------------------------*/
		for (register int xi=x_min;xi<x_max;xi++){
			for (register int yi=y_min;yi<y_max;yi++){
				int ImgIdx=xi+yi*width;
				int labels_t=dev_klabels[ImgIdx];
#if 1
				if (labels_t==SeedIdx){
					dev_sigmal[SeedIdx] += dev_m_lvec[ImgIdx];
					dev_sigmaa[SeedIdx] += dev_m_avec[ImgIdx];
					dev_sigmab[SeedIdx] += dev_m_bvec[ImgIdx];
					dev_sigmax[SeedIdx] += xi;
					dev_sigmay[SeedIdx] += yi;
					dev_clustersize[SeedIdx] += 1.0;
				}
#endif
			}
		}
		/*--------------------------------------*/
		SeedIdx+=offset;
	}
}
/*------------------------------------------------------------------------------------------*/
/**
*�����µ�ͼ�������   
*
*
*@param		    	   kseedSize			��������	
*@param				    m_width				ͼ����	
*@param				    m_height			ͼ��߶�	
*@param		[in]		dev_sigmal			��L����ͳ�����
*@param		[in]		dev_sigmaa			ͼ��A��������
*@param		[in]		dev_sigmab			ͼ��B��������
*@param		[in]		dev_sigmax			��X����ͳ�����
*@param		[in]		dev_sigmay			��Y����ͳ�����
*@param		[in]		dev_clustersize		������ͼ��������С
*@param		[out]		dev_kseedsl		    ����L����
*@param		[out]		dev_kseedsa		    ͼ��A����
*@param		[out]		dev_kseedsb		    ͼ��B����
*@param		[out]		dev_kseedsx		    ����X����
*@param		[out]		dev_kseedsy		    ����Y����
*@param		[out]		dev_kseed_sita_n	ͼ��ĽǶȷ���
*@param		[out]		dev_kseed_m_n	    ͼ���ģ������
*@param		[out]		dev_kseed_L_n	    ͼ������ȷ���
*@param		[out]		dev_kseed_X_n       ͼ���X�������
*@param		[out]		dev_kseed_Y_n	    ͼ���Y�������
*
*
*/
/*------------------------------------------------------------------------------------------*/
__global__ void StatisticsLABXY_Seed_calculate(
	const int			kseedSize,
	const int			m_width,
	const int			m_height,
	double*				dev_sigmal,
	double*				dev_sigmaa,
	double*				dev_sigmab,
	double*				dev_sigmax,
	double*				dev_sigmay,
	double*				dev_clustersize,
	double*				dev_kseedsl,
	double*				dev_kseedsa,
	double*				dev_kseedsb,
	double*				dev_kseedsx,
	double*				dev_kseedsy,
	double*				dev_kseed_sita_n,
	double*				dev_kseed_m_n,
	double*				dev_kseed_L_n,
	double*				dev_kseed_X_n,
	double*				dev_kseed_Y_n)
{
	int ki=blockIdx.x*blockDim.x+threadIdx.x;
	int offset=blockDim.x*gridDim.x;
	while(ki<kseedSize){

		{
				if (dev_clustersize[ki]<=0)  dev_clustersize[ki]=1;
				double inv_k=1.0/dev_clustersize[ki];
				/////////////////////////////////////////////////
				assert(dev_clustersize[ki]>=0&&dev_clustersize[ki]<=m_width*m_height);
				///////////////////////////////////////////////////
				dev_kseedsl[ki] = dev_sigmal[ki]*inv_k;
				dev_kseedsa[ki] = dev_sigmaa[ki]*inv_k;
				dev_kseedsb[ki] = dev_sigmab[ki]*inv_k;
				dev_kseedsx[ki] = dev_sigmax[ki]*inv_k;
				dev_kseedsy[ki] = dev_sigmay[ki]*inv_k;
				///////////////////////////////////////////////////
#if 1
					ConvertLab2oml(
					m_width,
					m_height,
					dev_kseedsl[ki],
					dev_kseedsa[ki],
					dev_kseedsb[ki],
					dev_kseedsx[ki],
					dev_kseedsy[ki],
					&dev_kseed_sita_n[ki],
					&dev_kseed_m_n[ki],
					&dev_kseed_L_n[ki],
					&dev_kseed_X_n[ki],
					&dev_kseed_Y_n[ki]);
#endif
				
		}
		ki+=offset;

	}
	
}
/*------------------------------------------------------------------------------------------*/
/**
*
*
*@param			kseedsSize    ���Ӹ���
*@param			m_width       ͼ����
*@param			m_height      ͼ��߶�
*@param			dev_kseedsl   ͼ���L����
*@param			dev_kseedsa   ͼ���A����
*@param			dev_kseedsb   ͼ���B����
*@param			dev_kseedsx   ͼ���X�������
*@param			dev_kseedsy   ͼ���X�������
*@param			dev_kseeds_sita_n  ͼ��ĽǶȷ���
*@param			dev_kseeds_m_n     ͼ���ģ������
*@param			dev_kseeds_L_n     ͼ������ȷ���
*@param			dev_kseeds_X_n     ͼ���X�������
*@param			dev_kseeds_Y_n     ͼ���Y�������
*
*/
/*------------------------------------------------------------------------------------------*/
__global__ void PerformSuperpixelSLIC_Init_ColorConvert(
	const int			kseedsSize,
	const int			m_width,
	const int			m_height,
	double*				dev_kseedsl,
	double*				dev_kseedsa,
	double*				dev_kseedsb,
	double*				dev_kseedsx,
	double*				dev_kseedsy,
	double*				dev_kseeds_sita_n,
	double*				dev_kseeds_m_n,
	double*				dev_kseeds_L_n,
	double*				dev_kseeds_X_n,
	double*				dev_kseeds_Y_n)
{
	///////////////////////////////////////////
	int ind=blockIdx.x*blockDim.x+threadIdx.x;
	int step=blockDim.x*gridDim.x;
	//////////////////////////////////////////
	while(ind<kseedsSize){
			/////////////////////////////////////
#if 1
			ConvertLab2oml(
				m_width,
				m_height,
				dev_kseedsl[ind],
				dev_kseedsa[ind],
				dev_kseedsb[ind],
				dev_kseedsx[ind],
				dev_kseedsy[ind],
				&dev_kseeds_sita_n[ind],
				&dev_kseeds_m_n[ind],
				&dev_kseeds_L_n[ind],
				&dev_kseeds_X_n[ind],
				&dev_kseeds_Y_n[ind]);
#endif
			ind+=step;
	}
	///////////////////////////////////////
}
/*------------------------------------------------------------------------------------------*/
/**
*
*��LABɫ��װ�����Ƕȡ�ģ�������� ��X��Y��ɵķ���
*
*@param   kseedsSize  ���Ӹ���
*@param   m_width     ͼ����
*@param   m_height    ͼ��߶�
*@param  [in]  dev_m_lvec  ͼ���L����
*@param  [in] dev_m_avec  ͼ���A����
*@param  [in] dev_m_bvec  ͼ���B����
*@param  [out] dev_sita_n  ͼ��ĽǶȷ���
*@param  [out] dev_m_n	  ͼ���ģ������
*@param  [out] dev_L_n     ͼ������ȷ���
*@param  [out] dev_X_n     ͼ���X�������
*@param  [out] dev_Y_n     ͼ���Y�������
*
*/
/*------------------------------------------------------------------------------------------*/
__global__ void PerformSuperpixelSLIC_ColorConvert(
	const int kseedsSize,
	const int m_width,
	const int m_height,
	double*				dev_m_lvec,
	double*				dev_m_avec,
	double*				dev_m_bvec,
	double*				dev_sita_n,
	double*				dev_m_n,
	double*				dev_L_n,
	double*				dev_X_n,
	double*				dev_Y_n)
{
	///////////////////////////////////////////
	int xi=blockIdx.x*blockDim.x+threadIdx.x;
	int yj=blockIdx.y*blockDim.y+threadIdx.y;
	int ind=xi+yj*m_width;
	//////////////////////////////////////////
	if(xi<m_width&&
		xi>=0&&
		yj>=0&&
		yj<m_height){
		/////////////////////////////////////
#if 1
			ConvertLab2oml(
				m_width,
				m_height,
				dev_m_lvec[ind],
				dev_m_avec[ind],
				dev_m_bvec[ind],
				xi,
				yj,
				&dev_sita_n[ind],
				&dev_m_n[ind],
				&dev_L_n[ind],
				&dev_X_n[ind],
				&dev_Y_n[ind]);
#endif

		///////////////////////////////////
	}
	///////////////////////////////////////
}
/*------------------------------------------------------------------------------------------*/
/**
*��LABɫ��װ�����Ƕȡ�ģ�������� ��X��Y��ɵķ���
*
*@param   kseedsSize  ���Ӹ���
*@param   m_width     ͼ����
*@param   m_height    ͼ��߶�
*@param  [in]  dev_m_lvec  ͼ���L����
*@param  [in] dev_m_avec  ͼ���A����
*@param  [in] dev_m_bvec  ͼ���B����
*@param  [out] dev_sita_n  ͼ��ĽǶȷ���
*@param  [out] dev_m_n	  ͼ���ģ������
*@param  [out] dev_L_n     ͼ������ȷ���
*@param  [out] dev_X_n     ͼ���X�������
*@param  [out] dev_Y_n     ͼ���Y�������
*
*
*/
/*------------------------------------------------------------------------------------------*/
void PerformSuperpixelSLIC_ColorConvert_gpu(
	const int kseedsSize,
	const int m_width,
	const int m_height,
	double*				dev_m_lvec,
	double*				dev_m_avec,
	double*				dev_m_bvec,
	double*				dev_sita_n,
	double*				dev_m_n,
	double*				dev_L_n,
	double*				dev_X_n,
	double*				dev_Y_n)
{
#if  _DEBUG
	hipEvent_t start,stop;
	hipError_t cudaStatus;
	float costtime_ms=0;
	float costtime_us=0;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	///////////////////////////////////////////////////////////////////////////////////////
	hipEventRecord(start,0);
#endif
	{
		dim3    threadsPerBlock(16,16);
		dim3    numBlock((m_width+threadsPerBlock.x-1)/threadsPerBlock.x,(m_height+threadsPerBlock.y-1)/threadsPerBlock.y);
		PerformSuperpixelSLIC_ColorConvert<<<numBlock,threadsPerBlock>>>(
			kseedsSize,
			m_width,
			m_height,
			dev_m_lvec,
			dev_m_avec,
			dev_m_bvec,
			dev_sita_n,
			dev_m_n,
			dev_L_n,
			dev_X_n,
			dev_Y_n);
	}
	////////////////////////////////////////////////////////////////////////////////////////
	cudaGetLastError_Sync_CUI();
#if _DEBUG
	cudaStatus=hipEventRecord(stop,0);	
	cudaGetLastError_Sync_CUI();
	cudaStatus=hipEventElapsedTime(&costtime_ms,start,stop);	
	costtime_us=costtime_ms*1000;
#endif
	cudaGetLastError_Sync_CUI();
#if _DEBUG
	hipEventDestroy(start);
	hipEventDestroy(stop);
#endif

}
/*------------------------------------------------------------------------------------------*/
/**
*����GPUʵ�ֳ������㷨
*
*@param			alpha			�Ƕ�ϵ��
*@param			betta			ģ��ϵ��
*@param			gama			����ϵ��
*@param			fai				�������
*@param			kseedsSize		���Ӹ��� 
*@param			m_width			ͼ����
*@param			m_height		ͼ��߶�
*@param		[]	host_m_lvec		ͼ��L����
*@param		[]	host_m_avec		ͼ��A����
*@param		[]	host_m_bvec		ͼ��B����
*@param		[]	host_kseedsl	���ӵ�L����
*@param		[]	host_kseedsa	���ӵ�A����
*@param		[]	host_kseedsb	���ӵ�B����
*@param		[]	host_kseedsx	���ӵ�X�������
*@param		[]	host_kseedsy	���ӵ�Y�������
*@param		[]	host_klabels	ͼ��ı궨��Ϣ
*@param			STEP
*@param		[]	host_edgemag
*@param  		M
*
*/
/*------------------------------------------------------------------------------------------*/
void PerformSuperpixelSLIC_gpu(
	float alpha,
	float betta,
	float gama,
	float fai,
	const int kseedsSize,
	const int m_width,
	const int m_height,
	double*				host_m_lvec,
	double*				host_m_avec,
	double*				host_m_bvec,
	double*				host_kseedsl,
	double*				host_kseedsa,
	double*				host_kseedsb,
	double*				host_kseedsx,
	double*				host_kseedsy,
	int*				host_klabels,
	const int			STEP,
	double*				host_edgemag,
	const double		M)
{
	hipError_t cudaStatus;
	///////////////////////////////////////////
	double*				dev_kseedsl;
	double*				dev_kseedsa;
	double*				dev_kseedsb;
	double*				dev_kseedsx;
	double*				dev_kseedsy;
	int*				dev_klabels;
	double*				dev_m_lvec;
	double*				dev_m_avec;
	double*				dev_m_bvec;
	double*				dev_sita_n;
	double*				dev_m_n;
	double*				dev_L_n;
	double*				dev_X_n;
	double*				dev_Y_n;
	double*				dev_edgemag=nullptr;
	///////////////////////////////////////////
	double* dev_distvec;
	///////////////////////////////////////////
	assert(cudaInit_CUI()==true);

	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop,0);
	int blocks=prop.multiProcessorCount;

	{
		cudaStatus = hipMalloc((void**)&dev_kseedsl, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_kseedsa, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_kseedsb, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_kseedsx, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_kseedsy, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_klabels,m_width*m_height * sizeof(int));
		cudaStatus = hipMalloc((void**)&dev_m_lvec,m_width*m_height * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_m_avec,m_width*m_height * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_m_bvec,m_width*m_height * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_distvec, m_width*m_height* sizeof(double));
		////////////////////////////////////////////////////////////////////////////////
		cudaStatus = hipMalloc((void**)&dev_sita_n, m_width*m_height* sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_m_n, m_width*m_height* sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_L_n, m_width*m_height* sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_X_n, m_width*m_height* sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_Y_n, m_width*m_height* sizeof(double));
		////////////////////////////////////////////////////////////////////////////////
		cudaStatus = hipMemcpy(dev_kseedsl, host_kseedsl, kseedsSize * sizeof(double), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(dev_kseedsa, host_kseedsa, kseedsSize * sizeof(double), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(dev_kseedsb, host_kseedsb, kseedsSize * sizeof(double), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(dev_kseedsx, host_kseedsx, kseedsSize * sizeof(double), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(dev_kseedsy, host_kseedsy, kseedsSize * sizeof(double), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(dev_klabels, host_klabels, m_width*m_height * sizeof(int), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(dev_m_lvec, host_m_lvec, m_width*m_height * sizeof(double), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(dev_m_avec, host_m_avec, m_width*m_height * sizeof(double), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(dev_m_bvec, host_m_bvec, m_width*m_height * sizeof(double), hipMemcpyHostToDevice);
		////////////////////////////////////////////////////////////////////////////////
	}
	double* dev_sigmal;
	double* dev_sigmaa;
	double* dev_sigmab;
	double* dev_sigmax;
	double* dev_sigmay;
	double* dev_clustersize;
	/////////////////////////////////////////////////////////////////////////////////////
	double*  dev_kseed_sita_n;
	double*  dev_kseed_m_n;
	double*  dev_kseed_L_n;
	double*  dev_kseed_X_n;
	double*  dev_kseed_Y_n;
	/////////////////////////////////////////////////////////////////////////////////////
	{
		cudaStatus = hipMalloc((void**)&dev_sigmal, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_sigmaa, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_sigmab, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_sigmax, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_sigmay, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_clustersize, kseedsSize * sizeof(double));
		cudaStatus = hipMemset(dev_clustersize,0,kseedsSize*sizeof(double));
		cudaGetLastError_Sync_CUI();
		////////////////////////////////////////////////////////////////////////////////
		cudaStatus = hipMalloc((void**)&dev_kseed_sita_n, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_kseed_m_n, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_kseed_L_n, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_kseed_X_n, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_kseed_Y_n, kseedsSize * sizeof(double));
	}
	hipEvent_t start,stop;
	float costtime_ms=0;
	float costtime_us=0;
	hipEventCreate(&start);
	hipEventCreate(&stop);
#if 1
	///////////////////////////////////////////
	PerformSuperpixelSLIC_ColorConvert_gpu(
		kseedsSize,
		m_width,
		m_height,
		dev_m_lvec,
		dev_m_avec,
		dev_m_bvec,
		dev_sita_n,
		dev_m_n,
		dev_L_n,
		dev_X_n,
		dev_Y_n);
	//////////////////////////////////////////
	PerformSuperpixelSLIC_Init_ColorConvert<<<1,256>>>(
		kseedsSize,
		m_width,
		m_height,
		dev_kseedsl,
		dev_kseedsa,
		dev_kseedsb,
		dev_kseedsx,
		dev_kseedsy,
		dev_kseed_sita_n,
		dev_kseed_m_n,
		dev_kseed_L_n,
		dev_kseed_X_n,
		dev_kseed_Y_n);
	//////////////////////////////////////////
#endif

	for( int itr = 0; itr < 10; itr++ )
	{
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		
		{
			cudaStatus=hipEventRecord(start,0);
#if 0
			dim3    threadsPerBlock(16,16);
			dim3    numBlock((m_width+15)/threadsPerBlock.x,(m_height+15)/threadsPerBlock.y);
			SetMatrix<<<numBlock,threadsPerBlock>>>(dev_distvec,m_width,m_height,1.7976931348623158e+308);
#else
			SetMatrix_Efficient<<<16,16*16>>>(dev_distvec,m_width,m_height,1.7976931348623158e+308);
#endif
			
			cudaGetLastError_Sync_CUI();
			cudaStatus=hipEventRecord(stop,0);
			cudaGetLastError_Sync_CUI();
			cudaStatus=hipEventElapsedTime(&costtime_ms,start,stop);
			costtime_us=costtime_ms*1000;
		}
		
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		{
			cudaStatus=hipEventRecord(start,0);
#if 0
			SetMinimumOnLab<<<(kseedsSize+511)/512,512>>>(
				kseedsSize,
				alpha,
				betta,
				gama,
				fai,
				m_width,
				m_height,
				dev_m_lvec,
				dev_m_avec,
				dev_m_bvec,
				dev_kseedsl,
				dev_kseedsa,
				dev_kseedsb,
				dev_kseedsx,
				dev_kseedsy,
				dev_klabels,
				dev_distvec,
				STEP,
				M);
#else
			dim3    threadsPerBlock(32,32);
			dim3    numBlock((m_width+31)/threadsPerBlock.x,(m_height+31)/threadsPerBlock.y);
			SetMinimumOnLab_AllSeeds<<<numBlock,threadsPerBlock>>>(
				kseedsSize,
				alpha,
				betta,
				gama,
				fai,
				m_width,
				m_height,
				dev_m_lvec,
				dev_m_avec,
				dev_m_bvec,
				dev_sita_n,
				dev_m_n,
				dev_L_n,
				dev_X_n,
				dev_Y_n,
				dev_kseedsl,
				dev_kseedsa,
				dev_kseedsb,
				dev_kseedsx,
				dev_kseedsy,
				dev_kseed_sita_n,
				dev_kseed_m_n,
				dev_kseed_L_n,
				dev_kseed_X_n,
				dev_kseed_Y_n,
				dev_klabels,
				dev_distvec);
#endif
			cudaGetLastError_Sync_CUI();
			cudaStatus=hipEventRecord(stop,0);	
			cudaGetLastError_Sync_CUI();
			cudaStatus=hipEventElapsedTime(&costtime_ms,start,stop);	
			costtime_us=costtime_ms*1000;
		}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		{
			/*cudaStatus=hipEventRecord(start,0);*/
			/*---------------------------------------------------------------------------------*/
//			const int numk=kseedsSize;
			hipMemset(dev_sigmal,0,kseedsSize*sizeof(double));
			cudaGetLastError_Sync_CUI();
			hipMemset(dev_sigmaa,0,kseedsSize*sizeof(double));
			cudaGetLastError_Sync_CUI();
			hipMemset(dev_sigmab,0,kseedsSize*sizeof(double));
			cudaGetLastError_Sync_CUI();
			hipMemset(dev_sigmax,0,kseedsSize*sizeof(double));
			cudaGetLastError_Sync_CUI();
			hipMemset(dev_sigmay,0,kseedsSize*sizeof(double));
			cudaGetLastError_Sync_CUI();
			hipMemset(dev_clustersize,0,kseedsSize*sizeof(double));
			cudaGetLastError_Sync_CUI();
			/*----------------------------------------------------------------------------------*/
			dim3    threadsPerBlock(16,16);
			dim3    numBlock((m_width+15)/threadsPerBlock.x,(m_height+15)/threadsPerBlock.y);
			unsigned int *mutex; 
			unsigned int zero = 0; 
			hipMalloc( (void**)&mutex, sizeof(unsigned int) ); 
			hipMemcpy(mutex, &zero, sizeof(unsigned int), hipMemcpyHostToDevice );
#if 0
			cudaStatus=hipEventRecord(start,0);
			//1225.2045ms
			StatisticsLABXY_Seed_collect<<<numBlock,threadsPerBlock>>>(
				mutex,
				kseedsSize,
				m_width,
				m_height,
				dev_m_lvec,
				dev_m_avec,
				dev_m_bvec,
				dev_klabels,
				dev_sigmal,
				dev_sigmaa,
				dev_sigmab,
				dev_sigmax,
				dev_sigmay,
				dev_clustersize);
			cudaGetLastError_Sync_CUI();
			cudaStatus=hipEventRecord(stop,0);	
			cudaGetLastError_Sync_CUI();
			cudaStatus=hipEventElapsedTime(&costtime_ms,start,stop);	
			costtime_us=costtime_ms*1000;
#endif
#if 0
			//69.114082ms
			StatisticsLABXY_Seed_collect_CPU(
				kseedsSize,
				m_width,
				m_height,
				host_m_lvec,
				host_m_avec,
				host_m_bvec,
				dev_klabels,
				dev_sigmal,
				dev_sigmaa,
				dev_sigmab,
				dev_sigmax,
				dev_sigmay,
				dev_clustersize);
#endif
#if 1
			cudaStatus=hipEventRecord(start,0);
			//344.65720ms
			StatisticsLABXY_Seed_collect_new2014_11_3<<<blocks*2,1024>>>(
				mutex,
				kseedsSize,
				m_width,
				m_height,
				dev_m_lvec,
				dev_m_avec,
				dev_m_bvec,
				dev_klabels,
				dev_sigmal,
				dev_sigmaa,
				dev_sigmab,
				dev_sigmax,
				dev_sigmay,
				dev_clustersize);
			cudaGetLastError_Sync_CUI();
			cudaStatus=hipEventRecord(stop,0);	
			cudaGetLastError_Sync_CUI();
			cudaStatus=hipEventElapsedTime(&costtime_ms,start,stop);	
			costtime_us=costtime_ms*1000;
#endif
#if 0
			cudaStatus=hipEventRecord(start,0);
			dim3    threadsPerBlock_Quad(32,32);
			dim3    numBlock_Quad(4,4);
			StatisticsLABXY_Seed_collect_blocks<<<numBlock_Quad,threadsPerBlock_Quad>>>(
				kseedsSize,
				m_width,
				m_height,
				dev_m_lvec,
				dev_m_avec,
				dev_m_bvec,
				dev_klabels,
				dev_sigmal,
				dev_sigmaa,
				dev_sigmab,
				dev_sigmax,
				dev_sigmay,
				dev_clustersize);
			cudaGetLastError_Sync_CUI();
			cudaStatus=hipEventRecord(stop,0);	
			cudaGetLastError_Sync_CUI();
			cudaStatus=hipEventElapsedTime(&costtime_ms,start,stop);	
			costtime_us=costtime_ms*1000;

#endif
			cudaStatus=hipEventRecord(start,0);
			cudaGetLastError_Sync_CUI();
			StatisticsLABXY_Seed_calculate<<<1,512>>>(
				kseedsSize,
				m_width,
				m_height,
				dev_sigmal,
				dev_sigmaa,
				dev_sigmab,
				dev_sigmax,
				dev_sigmay,
				dev_clustersize,
				dev_kseedsl,
				dev_kseedsa,
				dev_kseedsb,
				dev_kseedsx,
				dev_kseedsy,
				dev_kseed_sita_n,
				dev_kseed_m_n,
				dev_kseed_L_n,
				dev_kseed_X_n,
				dev_kseed_Y_n);
			cudaGetLastError_Sync_CUI();
			cudaStatus=hipEventRecord(stop,0);	
			cudaGetLastError_Sync_CUI();
			cudaStatus=hipEventElapsedTime(&costtime_ms,start,stop);	
			costtime_us=costtime_ms*1000;
		}
		
	}
	hipMemcpy(host_klabels,dev_klabels,m_width*m_height*sizeof(int),hipMemcpyDeviceToHost);
	{
	hipFree(dev_kseedsl);
	hipFree(dev_kseedsa);
	hipFree(dev_kseedsb);
	hipFree(dev_kseedsx);
	hipFree(dev_kseedsy);
	hipFree(dev_klabels);
	hipFree(dev_m_lvec);
	hipFree(dev_m_avec);
	hipFree(dev_m_bvec);
	hipFree(dev_distvec);
	////////////////////////
	hipFree(dev_sita_n);
	hipFree(dev_m_n);
	hipFree(dev_L_n);
	hipFree(dev_X_n);
	hipFree(dev_Y_n);
		{
			hipFree(dev_sigmal);
			hipFree(dev_sigmaa);
			hipFree(dev_sigmab);
			hipFree(dev_sigmax);
			hipFree(dev_sigmay);
			hipFree(dev_clustersize);
			//////////////////////////
			hipFree(dev_kseed_sita_n) ;
			hipFree(dev_kseed_m_n);
			hipFree(dev_kseed_L_n);
			hipFree(dev_kseed_X_n);
			hipFree(dev_kseed_Y_n);
		}
	}
	hipEventDestroy(start);
	hipEventDestroy(stop);
	assert(cudaDeInit_CUI()==true);
	
}
/*------------------------------------------------------------------------------------------*/
/**
*cuda����ʵ�г������㷨��
*
*
*@param    alpha  �ǶȲ���alpha
*@param    betta  ģֵbetta
*@param    gama   ����gama
*@param    fai    XY����
*@param    kseedsSize  ���Ӹ���
*@param    m_width   ͼ����
*@param    m_height  ͼ��߶�
*@param	   m_lvec  ͼ��L����
*@param	   m_avec  ͼ��A����
*@param	   m_bvec  ͼ��B����
*@param	   kseedsl  ����L����
*@param	   kseedsa  ����A����
*@param	   kseedsb  ����B����
*@param	   kseedsx  ����X����
*@param	   kseedsy  ����Y����
*@param	   klabels  ͼ��ͼ��ı궨ͼ��
*@param	   STEP     δʹ��
*@param	   edgemag  δʹ��
*@param	   M        δʹ���ܶ�����
*
*
*@note
*����GPUʵ�ֳ������㷨
*Сͼ1.8S  ��ͼ7.2S
*
*/
/*------------------------------------------------------------------------------------------*/
void PerformSuperpixelSLIC_gpu_simplify(
	float alpha,
	float betta,
	float gama,
	float fai,
	const int kseedsSize,
	const int m_width,   
	const int m_height,
	double*				host_m_lvec,
	double*				host_m_avec,
	double*				host_m_bvec,
	double*				host_kseedsl,
	double*				host_kseedsa,
	double*				host_kseedsb,
	double*				host_kseedsx,
	double*				host_kseedsy,
	int*				host_klabels,
	const int			STEP,
	double*				host_edgemag,
	const double		M)
{
	hipError_t cudaStatus;
	///////////////////////////////////////////
	double*				dev_kseedsl;
	double*				dev_kseedsa;
	double*				dev_kseedsb;
	double*				dev_kseedsx;
	double*				dev_kseedsy;
	int*				dev_klabels;
	double*				dev_m_lvec;
	double*				dev_m_avec;
	double*				dev_m_bvec;
	double*				dev_sita_n;
	double*				dev_m_n;
	double*				dev_L_n;
	double*				dev_X_n;
	double*				dev_Y_n;
	double*				dev_edgemag=nullptr;
	///////////////////////////////////////////
	double* dev_distvec;
	///////////////////////////////////////////
	assert(cudaInit_CUI()==true);
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop,0);
	int blocks=prop.multiProcessorCount;
	{
		cudaStatus = hipMalloc((void**)&dev_kseedsl, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_kseedsa, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_kseedsb, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_kseedsx, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_kseedsy, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_klabels,m_width*m_height * sizeof(int));
		cudaStatus = hipMalloc((void**)&dev_m_lvec,m_width*m_height * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_m_avec,m_width*m_height * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_m_bvec,m_width*m_height * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_distvec, m_width*m_height* sizeof(double));
		////////////////////////////////////////////////////////////////////////////////
		cudaStatus = hipMalloc((void**)&dev_sita_n, m_width*m_height* sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_m_n, m_width*m_height* sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_L_n, m_width*m_height* sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_X_n, m_width*m_height* sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_Y_n, m_width*m_height* sizeof(double));
		////////////////////////////////////////////////////////////////////////////////
		cudaStatus = hipMemcpy(dev_kseedsl, host_kseedsl, kseedsSize * sizeof(double), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(dev_kseedsa, host_kseedsa, kseedsSize * sizeof(double), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(dev_kseedsb, host_kseedsb, kseedsSize * sizeof(double), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(dev_kseedsx, host_kseedsx, kseedsSize * sizeof(double), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(dev_kseedsy, host_kseedsy, kseedsSize * sizeof(double), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(dev_klabels, host_klabels, m_width*m_height * sizeof(int), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(dev_m_lvec, host_m_lvec, m_width*m_height * sizeof(double), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(dev_m_avec, host_m_avec, m_width*m_height * sizeof(double), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(dev_m_bvec, host_m_bvec, m_width*m_height * sizeof(double), hipMemcpyHostToDevice);
		////////////////////////////////////////////////////////////////////////////////
	}
	double* dev_sigmal;
	double* dev_sigmaa;
	double* dev_sigmab;
	double* dev_sigmax;
	double* dev_sigmay;
	double* dev_clustersize;
	/////////////////////////////////////////////////////////////////////////////////////
	double*  dev_kseed_sita_n;
	double*  dev_kseed_m_n;
	double*  dev_kseed_L_n;
	double*  dev_kseed_X_n;
	double*  dev_kseed_Y_n;
	/////////////////////////////////////////////////////////////////////////////////////
	{
		cudaStatus = hipMalloc((void**)&dev_sigmal, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_sigmaa, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_sigmab, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_sigmax, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_sigmay, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_clustersize, kseedsSize * sizeof(double));
		cudaStatus=hipMemset(dev_clustersize,0,kseedsSize*sizeof(double));
		cudaGetLastError_Sync_CUI();
		////////////////////////////////////////////////////////////////////////////////
		cudaStatus = hipMalloc((void**)&dev_kseed_sita_n, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_kseed_m_n, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_kseed_L_n, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_kseed_X_n, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_kseed_Y_n, kseedsSize * sizeof(double));
	}
	hipEvent_t start,stop;
	float costtime_ms=0;
	float costtime_us=0;
	hipEventCreate(&start);
	hipEventCreate(&stop);
#if 1
	///////////////////////////////////////////
	PerformSuperpixelSLIC_ColorConvert_gpu(
		kseedsSize,
		m_width,
		m_height,
		dev_m_lvec,
		dev_m_avec,
		dev_m_bvec,
		dev_sita_n,
		dev_m_n,
		dev_L_n,
		dev_X_n,
		dev_Y_n);
	//////////////////////////////////////////
	PerformSuperpixelSLIC_Init_ColorConvert<<<1,256>>>(
		kseedsSize,
		m_width,
		m_height,
		dev_kseedsl,
		dev_kseedsa,
		dev_kseedsb,
		dev_kseedsx,
		dev_kseedsy,
		dev_kseed_sita_n,
		dev_kseed_m_n,
		dev_kseed_L_n,
		dev_kseed_X_n,
		dev_kseed_Y_n);
	cudaGetLastError_Sync_CUI();
	//////////////////////////////////////////
#endif
	for( int itr = 0; itr <10; itr++ )
	{
		//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

		{
			cudaStatus=hipEventRecord(start,0);
			SetMatrix_Efficient<<<16,16*16>>>(dev_distvec,m_width,m_height,1.7976931348623158e+308);
			cudaGetLastError_Sync_CUI();
			cudaStatus=hipEventRecord(stop,0);
			cudaGetLastError_Sync_CUI();
			cudaStatus=hipEventElapsedTime(&costtime_ms,start,stop);
			costtime_us=costtime_ms*1000;
		}
		
		//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		{
			cudaStatus=hipEventRecord(start,0);
			dim3    threadsPerBlock(16,16);
			dim3    numBlock((m_width+threadsPerBlock.x-1)/threadsPerBlock.x,(m_height+threadsPerBlock.y-1)/threadsPerBlock.y);

			SetMinimumOnThetaMLXY_9Seeds<<<numBlock,threadsPerBlock>>>(		
					kseedsSize,
					alpha,
					betta,
					gama,
					fai,
					m_width,
					m_height,			
					dev_sita_n,
					dev_m_n,
					dev_L_n,
					dev_X_n,
					dev_Y_n,				
					dev_kseed_sita_n,
					dev_kseed_m_n,
					dev_kseed_L_n,
					dev_kseed_X_n,
					dev_kseed_Y_n,
					dev_klabels,
					dev_distvec);	
			cudaGetLastError_Sync_CUI();
			cudaStatus=hipEventRecord(stop,0);
			cudaGetLastError_Sync_CUI();
			cudaStatus=hipEventElapsedTime(&costtime_ms,start,stop);
			costtime_us=costtime_ms*1000;
		}

		//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		{			
#if 1
			cudaStatus=hipEventRecord(start,0);
			/*---------------------------------------------------------------------------------*/
			//			const int numk=kseedsSize;
			hipMemset(dev_sigmal,0,kseedsSize*sizeof(double));
			cudaGetLastError_Sync_CUI();
			hipMemset(dev_sigmaa,0,kseedsSize*sizeof(double));
			cudaGetLastError_Sync_CUI();
			hipMemset(dev_sigmab,0,kseedsSize*sizeof(double));
			cudaGetLastError_Sync_CUI();
			hipMemset(dev_sigmax,0,kseedsSize*sizeof(double));
			cudaGetLastError_Sync_CUI();
			hipMemset(dev_sigmay,0,kseedsSize*sizeof(double));
			cudaGetLastError_Sync_CUI();
			hipMemset(dev_clustersize,0,kseedsSize*sizeof(double));
			cudaGetLastError_Sync_CUI();
			/*----------------------------------------------------------------------------------*/
			dim3    threadsPerBlock(16,16);
			dim3    numBlock((m_width+15)/threadsPerBlock.x,(m_height+15)/threadsPerBlock.y);
			cudaStatus=hipEventRecord(start,0);
			cudaGetLastError_Sync_CUI();
//69.114082ms
			StatisticsLABXY_Seed_collect_CPU(
				kseedsSize,
				m_width,
				m_height,
				host_m_lvec,
				host_m_avec,
				host_m_bvec,
				dev_klabels,
				nullptr,
				dev_sigmal,
				dev_sigmaa,
				dev_sigmab,
				dev_sigmax,
				dev_sigmay,
				dev_clustersize);
			cudaGetLastError_Sync_CUI();
			cudaStatus=hipEventRecord(stop,0);	
			cudaGetLastError_Sync_CUI();
			cudaStatus=hipEventElapsedTime(&costtime_ms,start,stop);	
			costtime_us=costtime_ms*1000;
						cudaStatus=hipEventRecord(start,0);
			cudaGetLastError_Sync_CUI();
			StatisticsLABXY_Seed_calculate<<<1,512>>>(
				kseedsSize,
				m_width,
				m_height,
				dev_sigmal,
				dev_sigmaa,
				dev_sigmab,
				dev_sigmax,
				dev_sigmay,
				dev_clustersize,
				dev_kseedsl,
				dev_kseedsa,
				dev_kseedsb,
				dev_kseedsx,
				dev_kseedsy,
				dev_kseed_sita_n,
				dev_kseed_m_n,
				dev_kseed_L_n,
				dev_kseed_X_n,
				dev_kseed_Y_n);
			cudaGetLastError_Sync_CUI();
			cudaStatus=hipEventRecord(stop,0);	
			cudaGetLastError_Sync_CUI();
			cudaStatus=hipEventElapsedTime(&costtime_ms,start,stop);	
			costtime_us=costtime_ms*1000;
			
#else

			cudaStatus=hipEventRecord(start,0);
			cudaGetLastError_Sync_CUI();
			StatisticsLABXY_Seed_collect_gpu_search<<<(kseedsSize+511)/512,512>>>(
				kseedsSize,
				m_width,
				m_height,
				dev_kseedsx,
				dev_kseedsy,
				dev_m_lvec,
				dev_m_avec,
				dev_m_bvec,
				dev_klabels,
				dev_sigmal,
				dev_sigmaa,
				dev_sigmab,
				dev_sigmax,
				dev_sigmay,
				dev_clustersize);
			cudaGetLastError_Sync_CUI();
			cudaStatus=hipEventRecord(stop,0);	
			cudaGetLastError_Sync_CUI();
			cudaStatus=hipEventElapsedTime(&costtime_ms,start,stop);	
			costtime_us=costtime_ms*1000;
#endif
		}
	}
#if 1
	////////////////////////////////////////////////////////////////////////////////////////
	cudaStatus=hipMemcpy(host_klabels,dev_klabels,m_width*m_height*sizeof(int),hipMemcpyDeviceToHost);	
	cudaStatus=hipMemcpy(host_kseedsl,dev_kseedsl, kseedsSize*sizeof(double),hipMemcpyDeviceToHost);	
	cudaStatus=hipMemcpy(host_kseedsa,dev_kseedsa, kseedsSize*sizeof(double),hipMemcpyDeviceToHost);	
	cudaStatus=hipMemcpy(host_kseedsb,dev_kseedsb, kseedsSize*sizeof(double),hipMemcpyDeviceToHost);	
	cudaStatus=hipMemcpy(host_kseedsx,dev_kseedsx, kseedsSize*sizeof(double),hipMemcpyDeviceToHost);	
	cudaStatus=hipMemcpy(host_kseedsy,dev_kseedsy, kseedsSize*sizeof(double),hipMemcpyDeviceToHost);
#endif
	///////////////////////////////////////////////////////////////////////////////////////
	{
		hipFree(dev_kseedsl);
		hipFree(dev_kseedsa);
		hipFree(dev_kseedsb);
		hipFree(dev_kseedsx);
		hipFree(dev_kseedsy);
		hipFree(dev_klabels);
		hipFree(dev_m_lvec);
		hipFree(dev_m_avec);
		hipFree(dev_m_bvec);
		hipFree(dev_distvec);
		////////////////////////
		hipFree(dev_sita_n);
		hipFree(dev_m_n);
		hipFree(dev_L_n);
		hipFree(dev_X_n);
		hipFree(dev_Y_n);
		{
			hipFree(dev_sigmal);
			hipFree(dev_sigmaa);
			hipFree(dev_sigmab);
			hipFree(dev_sigmax);
			hipFree(dev_sigmay);
			hipFree(dev_clustersize);
			//////////////////////////
			hipFree(dev_kseed_sita_n) ;
			hipFree(dev_kseed_m_n);
			hipFree(dev_kseed_L_n);
			hipFree(dev_kseed_X_n);
			hipFree(dev_kseed_Y_n);
		}
	}
	hipEventDestroy(start);
	hipEventDestroy(stop);
	assert(cudaDeInit_CUI()==true);
}
/*------------------------------------------------------------------------------------------*/
/**
*cuda����ʵ�г������㷨��
*
*
*@param    alpha  �ǶȲ���alpha
*@param    betta  ģֵbetta
*@param    gama   ����gama
*@param    fai    XY����
*@param    kseedsSize  ���Ӹ���
*@param    m_width   ͼ����
*@param    m_height  ͼ��߶�
*@param	   m_lvec  ͼ��L����
*@param	   m_avec  ͼ��A����
*@param	   m_bvec  ͼ��B����
*@param	   kseedsl  ����L����
*@param	   kseedsa  ����A����
*@param	   kseedsb  ����B����
*@param	   kseedsx  ����X����
*@param	   kseedsy  ����Y����
*@param	   klabels  ͼ��ͼ��ı궨ͼ��
*@param	   STEP     δʹ��
*@param	   edgemag  δʹ��
*@param	   M        δʹ���ܶ�����
*
*
*@note
*����GPUʵ�ֳ������㷨
*Сͼ1.8S  ��ͼ7.2S
*
*/
/*------------------------------------------------------------------------------------------*/
void PerformSuperpixelSLIC_gpu_simplify2(
	float alpha,
	float betta,
	float gama,
	float fai,
	const int kseedsSize,
	const int m_width,   
	const int m_height,
	double*				host_m_lvec,
	double*				host_m_avec,
	double*				host_m_bvec,
	double*				host_kseedsl,
	double*				host_kseedsa,
	double*				host_kseedsb,
	double*				host_kseedsx,
	double*				host_kseedsy,
	int*				host_klabels,
	const int			STEP,
	double*				host_edgemag,
	const double		M)
{
	hipError_t cudaStatus;
	///////////////////////////////////////////
	double*				dev_kseedsl;
	double*				dev_kseedsa;
	double*				dev_kseedsb;
	double*				dev_kseedsx;
	double*				dev_kseedsy;
	int*				dev_klabels;
	double*				dev_m_lvec;
	double*				dev_m_avec;
	double*				dev_m_bvec;
	double*				dev_sita_n;
	double*				dev_m_n;
	double*				dev_L_n;
	double*				dev_X_n;
	double*				dev_Y_n;
	double*				dev_edgemag=nullptr;
	///////////////////////////////////////////
	double* dev_distvec;
	///////////////////////////////////////////
	assert(cudaInit_CUI()==true);

	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop,0);
	int blocks=prop.multiProcessorCount;

	{
		cudaStatus = hipMalloc((void**)&dev_kseedsl, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_kseedsa, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_kseedsb, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_kseedsx, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_kseedsy, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_klabels,m_width*m_height * sizeof(int));
		cudaStatus = hipMalloc((void**)&dev_m_lvec,m_width*m_height * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_m_avec,m_width*m_height * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_m_bvec,m_width*m_height * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_distvec, m_width*m_height* sizeof(double));
		////////////////////////////////////////////////////////////////////////////////
		cudaStatus = hipMalloc((void**)&dev_sita_n, m_width*m_height* sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_m_n, m_width*m_height* sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_L_n, m_width*m_height* sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_X_n, m_width*m_height* sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_Y_n, m_width*m_height* sizeof(double));
		////////////////////////////////////////////////////////////////////////////////
		cudaStatus = hipMemcpy(dev_kseedsl, host_kseedsl, kseedsSize * sizeof(double), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(dev_kseedsa, host_kseedsa, kseedsSize * sizeof(double), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(dev_kseedsb, host_kseedsb, kseedsSize * sizeof(double), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(dev_kseedsx, host_kseedsx, kseedsSize * sizeof(double), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(dev_kseedsy, host_kseedsy, kseedsSize * sizeof(double), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(dev_klabels, host_klabels, m_width*m_height * sizeof(int), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(dev_m_lvec, host_m_lvec, m_width*m_height * sizeof(double), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(dev_m_avec, host_m_avec, m_width*m_height * sizeof(double), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(dev_m_bvec, host_m_bvec, m_width*m_height * sizeof(double), hipMemcpyHostToDevice);
		////////////////////////////////////////////////////////////////////////////////
	}

	double* dev_clustersize;
	/////////////////////////////////////////////////////////////////////////////////////
	double*  dev_kseed_sita_n;
	double*  dev_kseed_m_n;
	double*  dev_kseed_L_n;
	double*  dev_kseed_X_n;
	double*  dev_kseed_Y_n;
	/////////////////////////////////////////////////////////////////////////////////////
	{

		cudaStatus = hipMalloc((void**)&dev_clustersize, kseedsSize * sizeof(double));
		cudaStatus=hipMemset(dev_clustersize,0,kseedsSize*sizeof(double));
		cudaGetLastError_Sync_CUI();
		////////////////////////////////////////////////////////////////////////////////
		cudaStatus = hipMalloc((void**)&dev_kseed_sita_n, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_kseed_m_n, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_kseed_L_n, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_kseed_X_n, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_kseed_Y_n, kseedsSize * sizeof(double));
	}
	hipEvent_t start,stop;
	float costtime_ms=0;
	float costtime_us=0;
	hipEventCreate(&start);
	hipEventCreate(&stop);
#if 1
	///////////////////////////////////////////
	PerformSuperpixelSLIC_ColorConvert_gpu(
		kseedsSize,
		m_width,
		m_height,
		dev_m_lvec,
		dev_m_avec,
		dev_m_bvec,
		dev_sita_n,
		dev_m_n,
		dev_L_n,
		dev_X_n,
		dev_Y_n);
	//////////////////////////////////////////
	PerformSuperpixelSLIC_Init_ColorConvert<<<1,256>>>(
		kseedsSize,
		m_width,
		m_height,
		dev_kseedsl,
		dev_kseedsa,
		dev_kseedsb,
		dev_kseedsx,
		dev_kseedsy,
		dev_kseed_sita_n,
		dev_kseed_m_n,
		dev_kseed_L_n,
		dev_kseed_X_n,
		dev_kseed_Y_n);
	cudaGetLastError_Sync_CUI();
	//////////////////////////////////////////
#endif
	for( int itr = 0; itr <2; itr++ )
	{
		//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

		{
			cudaStatus=hipEventRecord(start,0);
			SetMatrix_Efficient<<<16,16*16>>>(dev_distvec,m_width,m_height,1.7976931348623158e+308);
			cudaGetLastError_Sync_CUI();
			cudaStatus=hipEventRecord(stop,0);
			cudaGetLastError_Sync_CUI();
			cudaStatus=hipEventElapsedTime(&costtime_ms,start,stop);
			costtime_us=costtime_ms*1000;
		}
		
		//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		{
			cudaStatus=hipEventRecord(start,0);
			dim3    threadsPerBlock(16,16);
			dim3    numBlock((m_width+threadsPerBlock.x-1)/threadsPerBlock.x,(m_height+threadsPerBlock.y-1)/threadsPerBlock.y);

			SetMinimumOnThetaMLXY_9Seeds<<<numBlock,threadsPerBlock>>>(		
					kseedsSize,
					alpha,
					betta,
					gama,
					fai,
					m_width,
					m_height,			
					dev_sita_n,
					dev_m_n,
					dev_L_n,
					dev_X_n,
					dev_Y_n,				
					dev_kseed_sita_n,
					dev_kseed_m_n,
					dev_kseed_L_n,
					dev_kseed_X_n,
					dev_kseed_Y_n,
					dev_klabels,
					dev_distvec);	
			cudaGetLastError_Sync_CUI();
			cudaStatus=hipEventRecord(stop,0);
			cudaGetLastError_Sync_CUI();
			cudaStatus=hipEventElapsedTime(&costtime_ms,start,stop);
			costtime_us=costtime_ms*1000;
		}

		//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		{			

			StatisticsThetaMLXY_Seed_collect_CPU_Incompletion(
								kseedsSize,
									m_width,
									m_height,
									host_m_lvec,
									host_m_avec,
									host_m_bvec,
									dev_klabels,
								    NULL,
									dev_kseed_sita_n,
									dev_kseed_m_n,
									dev_kseed_L_n,
									dev_kseed_X_n,
									dev_kseed_Y_n);
		}
	}
#if 1
	////////////////////////////////////////////////////////////////////////////////////////
	cudaStatus=hipMemcpy(host_klabels,dev_klabels,m_width*m_height*sizeof(int),hipMemcpyDeviceToHost);	
	cudaStatus=hipMemcpy(host_kseedsl,dev_kseedsl, kseedsSize*sizeof(double),hipMemcpyDeviceToHost);	
	cudaStatus=hipMemcpy(host_kseedsa,dev_kseedsa, kseedsSize*sizeof(double),hipMemcpyDeviceToHost);	
	cudaStatus=hipMemcpy(host_kseedsb,dev_kseedsb, kseedsSize*sizeof(double),hipMemcpyDeviceToHost);	
	cudaStatus=hipMemcpy(host_kseedsx,dev_kseedsx, kseedsSize*sizeof(double),hipMemcpyDeviceToHost);	
	cudaStatus=hipMemcpy(host_kseedsy,dev_kseedsy, kseedsSize*sizeof(double),hipMemcpyDeviceToHost);
#endif
	///////////////////////////////////////////////////////////////////////////////////////
	{
		hipFree(dev_kseedsl);
		hipFree(dev_kseedsa);
		hipFree(dev_kseedsb);
		hipFree(dev_kseedsx);
		hipFree(dev_kseedsy);
		hipFree(dev_klabels);
		hipFree(dev_m_lvec);
		hipFree(dev_m_avec);
		hipFree(dev_m_bvec);
		hipFree(dev_distvec);
		////////////////////////
		hipFree(dev_sita_n);
		hipFree(dev_m_n);
		hipFree(dev_L_n);
		hipFree(dev_X_n);
		hipFree(dev_Y_n);
		{
			//////////////////////////
			hipFree(dev_kseed_sita_n) ;
			hipFree(dev_kseed_m_n);
			hipFree(dev_kseed_L_n);
			hipFree(dev_kseed_X_n);
			hipFree(dev_kseed_Y_n);
		}
	}
	hipEventDestroy(start);
	hipEventDestroy(stop);
	assert(cudaDeInit_CUI()==true);
}
/*------------------------------------------------------------------------------------------*/
/**
*cuda����ʵ�г������㷨��
*
*
*@param    alpha  �ǶȲ���alpha
*@param    betta  ģֵbetta
*@param    gama   ����gama
*@param    fai    XY����
*@param    kseedsSize  ���Ӹ���
*@param    m_width   ͼ����
*@param    m_height  ͼ��߶�

*@param	   klabels  ͼ��ͼ��ı궨ͼ��
*@param	   STEP     δʹ��
*@param	   edgemag  δʹ��
*@param	   M        δʹ���ܶ�����
*
*
*@note
*����GPUʵ�ֳ������㷨
*Сͼ1.8S  ��ͼ7.2S
*
*/
/*------------------------------------------------------------------------------------------*/
void PerformSuperpixelSLIC_ThetaMLXY_gpu_simplify(
	float alpha,
	float betta,
	float gama,
	float fai,
	const int kseedsSize,
	const int m_width,   
	const int m_height,
	double*     host_sita_n,
	double*     host_m_n,
	double*     host_L_n,
	double*     host_X_n,
	double*     host_Y_n,
	double*     host_kseedsTheta,
	double*     host_kseedsM,
	double*     host_kseedsL,
	double*     host_kseedsX,
	double*     host_kseedsY,
	int*		host_klabels,
	const int			STEP)
{
	hipError_t cudaStatus;

	double*  dev_kseed_sita_n;
	double*  dev_kseed_m_n;
	double*  dev_kseed_L_n;
	double*  dev_kseed_X_n;
	double*  dev_kseed_Y_n;	
	double*	 dev_sita_n;
	double*	 dev_m_n;
	double*	 dev_L_n;
	double*	 dev_X_n;
	double*	 dev_Y_n;
	int*     dev_klabels;
	double*  dev_distvec;
	assert(cudaInit_CUI()==true);
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop,0);
		////////////////////////////////////////////////////////////////////////////////
		cudaStatus = hipMalloc((void**)&dev_klabels,m_width*m_height * sizeof(int));
		cudaStatus = hipMalloc((void**)&dev_distvec, m_width*m_height* sizeof(double));

		cudaStatus = hipMalloc((void**)&dev_kseed_sita_n, kseedsSize* sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_kseed_m_n, kseedsSize* sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_kseed_L_n, kseedsSize* sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_kseed_X_n, kseedsSize* sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_kseed_Y_n, kseedsSize* sizeof(double));

		cudaStatus = hipMalloc((void**)&dev_sita_n, m_width*m_height* sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_m_n, m_width*m_height* sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_L_n, m_width*m_height* sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_X_n, m_width*m_height* sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_Y_n, m_width*m_height* sizeof(double));
		////////////////////////////////////////////////////////////////////////////////
		cudaStatus = hipMemcpy(dev_klabels, host_klabels, m_width*m_height * sizeof(int), hipMemcpyHostToDevice);
		
		cudaStatus = hipMemcpy(dev_kseed_sita_n, host_kseedsTheta, kseedsSize * sizeof(double), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(dev_kseed_m_n, host_kseedsM, kseedsSize * sizeof(double), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(dev_kseed_L_n, host_kseedsL, kseedsSize * sizeof(double), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(dev_kseed_X_n, host_kseedsX, kseedsSize * sizeof(double), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(dev_kseed_Y_n, host_kseedsY, kseedsSize * sizeof(double), hipMemcpyHostToDevice);

		cudaStatus = hipMemcpy(dev_sita_n, host_sita_n, m_width*m_height * sizeof(double), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(dev_m_n, host_m_n,m_width*m_height * sizeof(double), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(dev_L_n, host_L_n,m_width*m_height * sizeof(double), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(dev_X_n, host_X_n,m_width*m_height * sizeof(double), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(dev_Y_n, host_Y_n,m_width*m_height * sizeof(double), hipMemcpyHostToDevice);
	hipEvent_t start,stop;
	float costtime_ms=0;
	float costtime_us=0;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	for( int itr = 0; itr <10; itr++ )
	{
		{
			cudaStatus=hipEventRecord(start,0);
			SetMatrix_Efficient<<<16,16*16>>>(dev_distvec,m_width,m_height,1.7976931348623158e+308);
			cudaGetLastError_Sync_CUI();
			cudaStatus=hipEventRecord(stop,0);
			cudaGetLastError_Sync_CUI();
			cudaStatus=hipEventElapsedTime(&costtime_ms,start,stop);
			costtime_us=costtime_ms*1000;
		}
			dim3    threadsPerBlock(16,16);
			dim3    numBlock((m_width+threadsPerBlock.x-1)/threadsPerBlock.x,(m_height+threadsPerBlock.y-1)/threadsPerBlock.y);
#if 1
			SetMinimumOnThetaMLXY_9Seeds<<<numBlock,threadsPerBlock>>>(			
							kseedsSize,
							alpha,
							betta,
							gama,
							fai,
							m_width,
							m_height,			
							dev_sita_n,
							dev_m_n,
							dev_L_n,
							dev_X_n,
							dev_Y_n,				
							dev_kseed_sita_n,
							dev_kseed_m_n,
							dev_kseed_L_n,
							dev_kseed_X_n,
							dev_kseed_Y_n,
							dev_klabels,
							dev_distvec);

			cudaGetLastError_Sync_CUI();

			StatisticsThetaMLXY_Seed_collect_CPU(
							kseedsSize,
							m_width,
							m_height,
							host_sita_n,
							host_m_n,
							host_L_n,
							host_X_n,
							host_Y_n,	
							dev_klabels,
    						dev_kseed_sita_n,
							dev_kseed_m_n,
							dev_kseed_L_n,
							dev_kseed_X_n,
							dev_kseed_Y_n);
#endif
			cudaGetLastError_Sync_CUI();
			cudaStatus=hipEventRecord(stop,0);
			cudaGetLastError_Sync_CUI();
			cudaStatus=hipEventElapsedTime(&costtime_ms,start,stop);
			costtime_us=costtime_ms*1000;
}	
		hipMemcpy(host_klabels,dev_klabels,m_width*m_height*sizeof(int),hipMemcpyDeviceToHost);		
		hipMemcpy(host_kseedsTheta,dev_kseed_sita_n,kseedsSize * sizeof(double),hipMemcpyDeviceToHost);		
		hipMemcpy(host_kseedsM,dev_kseed_m_n,kseedsSize * sizeof(double),hipMemcpyDeviceToHost);		
		hipMemcpy(host_kseedsL,dev_kseed_L_n,kseedsSize * sizeof(double),hipMemcpyDeviceToHost);		
		hipMemcpy(host_kseedsX,dev_kseed_X_n,kseedsSize * sizeof(double),hipMemcpyDeviceToHost);		
		hipMemcpy(host_kseedsY,dev_kseed_Y_n,kseedsSize * sizeof(double),hipMemcpyDeviceToHost);

	    hipFree(dev_sita_n);
		hipFree(dev_m_n);
		hipFree(dev_L_n);
		hipFree(dev_X_n);
		hipFree(dev_Y_n);
		hipFree(dev_kseed_sita_n) ;
		hipFree(dev_kseed_m_n);
		hipFree(dev_kseed_L_n);
		hipFree(dev_kseed_X_n);
		hipFree(dev_kseed_Y_n);
		hipFree(dev_distvec);
		hipFree(dev_klabels);
	assert(cudaDeInit_CUI()==true);
}
/*------------------------------------------------------------------------------------------*/
/**
*���������˺���
*
*@param		[in,out]	ubuff  ͼ������
*@param		[in]		labels ͼ��labels
*@param					width  ͼ����
*@param					height ͼ��߶�
*@param					color_in   ��ɫ�ڱ߽���ɫ
*@param					color_out  ��ɫ��߽���ɫ
*
*/
/*------------------------------------------------------------------------------------------*/
__global__ void DrawContoursAroundSegments_thread(
	unsigned int*			ubuff,
	int*					labels,
	const int				width,
	const int				height,
	const unsigned int		color_in,
	const unsigned int      color_out) 
{
	int k =blockIdx.x*blockDim.x+threadIdx.x;
	int j =blockIdx.y*blockDim.y+threadIdx.y;
	int mainindex = j*width + k;
	const int dx8[8] = {-1, -1,  0,  1, 1, 1, 0, -1};
	const int dy8[8] = { 0, -1, -1, -1, 0, 1, 1,  1};
	////////////////////////////////////////////////
		int np(0);
		if (k>=0&&k<width&&j>=0&&j<height){
				for(register int i = 0; i < 8; i++ ){
					int x = k + dx8[i];
					int y = j + dy8[i];
					if( (x >= 0 && x < width) && (y >= 0 && y < height) ){
								int index = y*width + x;
								//if( false == istaken[index] )//comment this to obtain internal contours
								{
									if( labels[mainindex] != labels[index] ){
										np++;
										//ubuff[index] =0xff000000; //2pixel����
										ubuff[index] =color_out; //2pixel����
									}
								}
					}
				}
				if( np > 1 ){
						ubuff[mainindex] =color_in;//2pixel����
				}
		}		
}
/*------------------------------------------------------------------------------------------*/
/**
*����GPU��������������ͼ����
*@param	  [in,out] ubuff     ͼ��
*@param	  [in] labels    labels����
*@param	   width     ͼ����
*@param	   height    ͼ��߶�
*@param	   color_in  �����ڱ߽���ɫ
*@param    color_out ������߽���ɫ
*
*/
/*------------------------------------------------------------------------------------------*/
void DrawContoursAroundSegments_gpu(
	unsigned int*			ubuff,
	int*					labels,
	const int				width,
	const int				height,
	const unsigned int	color_in,
	const unsigned int  color_out)
{
	unsigned int* ubuff_dev;
	int*  labels_dev;
	hipError_t cudaStatus;
	assert(cudaInit_CUI()==true);
	hipStream_t stream;//���Ľṹ��

	cudaStatus=hipStreamCreate(&stream);	
	cudaStatus=hipMalloc((void**)&ubuff_dev, width*height*sizeof(unsigned int));
	cudaStatus=hipMalloc((void**)&labels_dev,width*height*sizeof(int));

	cudaStatus = hipMemcpy(ubuff_dev,ubuff, width*height*sizeof(unsigned int), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(labels_dev,labels,width*height*sizeof(int), hipMemcpyHostToDevice);
	/////////////////////////////////////////////////////////////////////////////////////////
	dim3    threadsPerBlock(16,16);
	dim3    numBlock((width+threadsPerBlock.x-1)/threadsPerBlock.x,(height+threadsPerBlock.y-1)/threadsPerBlock.y);
	DrawContoursAroundSegments_thread<<<numBlock,threadsPerBlock>>>(ubuff_dev,
											labels_dev,
											width,
											height,
											color_in,
											color_out);
	cudaGetLastError_Sync_CUI();
	hipMemcpy(ubuff,ubuff_dev,width*height*sizeof(unsigned int),hipMemcpyDeviceToHost);

	cudaStatus=hipStreamDestroy(stream);
	hipFree(ubuff_dev);
	hipFree(labels_dev);
	/////////////////////////////////////////////////////////////////////////////////////////
	assert(cudaDeInit_CUI()==true);
}
/*------------------------------------------------------------------------------------------*/
/**
*Normal  0x00 00 00 00 
*MASK    0xff ff ff ff
*���ó����ص���������\n
*���ε�����
*@param
*@param	[out]	dev_mask      �����ε�����
*@param	[in]	dev_labels    labels����
*@param		width         ͼ����
*@param		height        ͼ��߶�
*@param     Pixel         ������
*/
/*------------------------------------------------------------------------------------------*/
void InitMaskTemplate(
	unsigned int*			dev_mask,
	int*					dev_labels,
	const int				width,
	const int				height,
	const int                Pixel)
{
	int color_in;
	int color_out;
	if (Pixel==2){
		color_in=0xffffffff;
		color_out=0x00000000;
	}else if(Pixel==4){		
		color_in=0xffffffff;
		color_out=0xffffffff;
	}
	
	hipMemset(dev_mask,0,sizeof(unsigned int)*width*height);
	dim3    threadsPerBlock(16,16);
	dim3    numBlock((width+threadsPerBlock.x-1)/threadsPerBlock.x,(height+threadsPerBlock.y-1)/threadsPerBlock.y);
	DrawContoursAroundSegments_thread<<<numBlock,threadsPerBlock>>>(dev_mask,
		dev_labels,
		width,
		height,
		color_in,
		color_out);
	cudaGetLastError_Sync_CUI();
}
/*------------------------------------------------------------------------------------------*/
/**
*
*����GPU������ȡ����
*.cu���ص���
*
*@param			 kseedsSize   ���Ӹ���
*@param			 m_width      ͼ����
*@param			 m_height     ͼ��߶�
*@param 	[in]	 host_m_lvec  ����L����
*@param 	[in]	 host_m_avec  ����A����
*@param 	[in]	 host_m_bvec  ����B����
*@param 	[out]	 host_kseedsl  ����L����
*@param 	[out]	 host_kseedsa  ����A����
*@param 	[out]	 host_kseedsb  ����B����
*@param 	[out]	 host_kseedsx  ����X����
*@param 	[out]	 host_kseedsy  ����Y����
*@param 	[in]	 host_klabels  ͼ��ͼ��ı궨ͼ��
*
*/
/*------------------------------------------------------------------------------------------*/
void  GetSeedsLabxy_gpu(
	const int kseedsSize,
	const int m_width,
	const int m_height,
	double*				host_m_lvec,
	double*				host_m_avec,
	double*				host_m_bvec,
	double*				host_kseedsl,
	double*				host_kseedsa,
	double*				host_kseedsb,
	double*				host_kseedsx,
	double*				host_kseedsy,
	int*				host_klabels)
{
	hipEvent_t start,stop;
	float costtime_ms=0;
	float costtime_us=0;
	hipError_t cudaStatus;
	unsigned int*       mask_img;
	double*				dev_kseedsl;
	double*				dev_kseedsa;
	double*				dev_kseedsb;
	double*				dev_kseedsx;
	double*				dev_kseedsy;
	int*				dev_klabels;
	double*			dev_m_lvec;
	double*			dev_m_avec;
	double*			dev_m_bvec;
	{
		hipEventCreate(&start);
		hipEventCreate(&stop);
		cudaStatus=hipMalloc((void**)&mask_img,sizeof(unsigned int)*m_width*m_height);
		cudaStatus = hipMalloc((void**)&dev_kseedsl, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_kseedsa, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_kseedsb, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_kseedsx, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_kseedsy, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_klabels,m_width*m_height * sizeof(int));
		cudaStatus = hipMalloc((void**)&dev_m_lvec,m_width*m_height * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_m_avec,m_width*m_height * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_m_bvec,m_width*m_height * sizeof(double));
#if 1
		cudaStatus = hipMemset(dev_kseedsl,0, kseedsSize * sizeof(double));
		cudaStatus = hipMemset(dev_kseedsa,0, kseedsSize * sizeof(double));
		cudaStatus = hipMemset(dev_kseedsb,0, kseedsSize * sizeof(double));
		cudaStatus = hipMemset(dev_kseedsx,0, kseedsSize * sizeof(double));
		cudaStatus = hipMemset(dev_kseedsy,0, kseedsSize * sizeof(double));
#endif
		cudaStatus = hipMemcpy(dev_klabels, host_klabels, m_width*m_height * sizeof(int), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(dev_m_lvec, host_m_lvec, m_width*m_height * sizeof(double), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(dev_m_avec, host_m_avec, m_width*m_height * sizeof(double), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(dev_m_bvec, host_m_bvec, m_width*m_height * sizeof(double), hipMemcpyHostToDevice);
	}
	
	double*			dev_sigmal;
	double*			dev_sigmaa;
	double*			dev_sigmab;
	double*			dev_sigmax;
	double*			dev_sigmay;
	double*			dev_clustersize;
	double*				dev_kseed_sita_n;
	double*				dev_kseed_m_n;
	double*				dev_kseed_L_n;
	double*				dev_kseed_X_n;
	double*			    dev_kseed_Y_n;
	{
		cudaStatus = hipMalloc((void**)&dev_sigmal, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_sigmaa, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_sigmab, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_sigmax, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_sigmay, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_clustersize, kseedsSize * sizeof(double));
		cudaStatus=hipMemset(dev_clustersize,0,kseedsSize*sizeof(double));
		cudaGetLastError_Sync_CUI();
		////////////////////////////////////////////////////////////////////////////////
		cudaStatus = hipMalloc((void**)&dev_kseed_sita_n, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_kseed_m_n, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_kseed_L_n, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_kseed_X_n, kseedsSize * sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_kseed_Y_n, kseedsSize * sizeof(double));
	}
		/*----------------------------------------------------------------------------------*/
			cudaStatus =hipMemset(dev_sigmal,0,kseedsSize*sizeof(double));			
			cudaStatus =hipMemset(dev_sigmaa,0,kseedsSize*sizeof(double));			
			cudaStatus =hipMemset(dev_sigmab,0,kseedsSize*sizeof(double));			
			cudaStatus =hipMemset(dev_sigmax,0,kseedsSize*sizeof(double));			
			cudaStatus =hipMemset(dev_sigmay,0,kseedsSize*sizeof(double));			
			cudaStatus =hipMemset(dev_clustersize,0,kseedsSize*sizeof(double));			
		/*----------------------------------------------------------------------------------*/

			InitMaskTemplate(mask_img,dev_klabels,m_width,m_height,4);

			dim3    threadsPerBlock(16,16);
			dim3    numBlock((m_width+15)/threadsPerBlock.x,(m_height+15)/threadsPerBlock.y);
			cudaStatus=hipEventRecord(start,0);
			cudaGetLastError_Sync_CUI();
#if 0
			StatisticsLABXY_Seed_collect_gpu_search_AllImg<<<(kseedsSize+511)/512,512>>>(
				kseedsSize,
				m_width,
				m_height,
				dev_m_lvec,
				dev_m_avec,
				dev_m_bvec,
				dev_klabels,
				dev_sigmal,
				dev_sigmaa,
				dev_sigmab,
				dev_sigmax,
				dev_sigmay,
				dev_clustersize);
#else
			StatisticsLABXY_Seed_collect_CPU(
				kseedsSize,
				m_width,
				m_height,
				host_m_lvec,
				host_m_avec,
				host_m_bvec,
				dev_klabels,
				mask_img,
				dev_sigmal,
				dev_sigmaa,
				dev_sigmab,
				dev_sigmax,
				dev_sigmay,
				dev_clustersize);
#endif
			
			cudaGetLastError_Sync_CUI();
			cudaStatus=hipEventRecord(stop,0);	
			cudaGetLastError_Sync_CUI();
			cudaStatus=hipEventElapsedTime(&costtime_ms,start,stop);	
			costtime_us=costtime_ms*1000;			
			cudaStatus=hipEventRecord(start,0);
			cudaGetLastError_Sync_CUI();
			StatisticsLABXY_Seed_calculate<<<1,512>>>(
				kseedsSize,
				m_width,
				m_height,
				dev_sigmal,
				dev_sigmaa,
				dev_sigmab,
				dev_sigmax,
				dev_sigmay,
				dev_clustersize,
				dev_kseedsl,
				dev_kseedsa,
				dev_kseedsb,
				dev_kseedsx,
				dev_kseedsy,
				dev_kseed_sita_n,
				dev_kseed_m_n,
				dev_kseed_L_n,
				dev_kseed_X_n,
				dev_kseed_Y_n);
			cudaGetLastError_Sync_CUI();
			cudaStatus=hipEventRecord(stop,0);	
			cudaGetLastError_Sync_CUI();
			cudaStatus=hipEventElapsedTime(&costtime_ms,start,stop);	
			costtime_us=costtime_ms*1000;
#if 1
		////////////////////////////////////////////////////////////////////////////////////////
		cudaStatus=hipMemcpy(host_klabels,dev_klabels,m_width*m_height*sizeof(int),hipMemcpyDeviceToHost);	
		cudaStatus=hipMemcpy(host_kseedsl,dev_kseedsl, kseedsSize*sizeof(double),hipMemcpyDeviceToHost);	
		cudaStatus=hipMemcpy(host_kseedsa,dev_kseedsa, kseedsSize*sizeof(double),hipMemcpyDeviceToHost);	
		cudaStatus=hipMemcpy(host_kseedsb,dev_kseedsb, kseedsSize*sizeof(double),hipMemcpyDeviceToHost);	
		cudaStatus=hipMemcpy(host_kseedsx,dev_kseedsx, kseedsSize*sizeof(double),hipMemcpyDeviceToHost);	
		cudaStatus=hipMemcpy(host_kseedsy,dev_kseedsy, kseedsSize*sizeof(double),hipMemcpyDeviceToHost);
#endif
		{
			hipEventDestroy(start);
			hipEventDestroy(stop);
			hipFree(mask_img);
			hipFree(dev_kseedsl);
			hipFree(dev_kseedsa);
			hipFree(dev_kseedsb);
			hipFree(dev_kseedsx);
			hipFree(dev_kseedsy);
			hipFree(dev_klabels);
			hipFree(dev_m_lvec);
			hipFree(dev_m_avec);
			hipFree(dev_m_bvec);
		/*	hipFree(dev_distvec);*/
			////////////////////////
			/*hipFree(dev_sita_n);
			hipFree(dev_m_n);
			hipFree(dev_L_n);
			hipFree(dev_X_n);
			hipFree(dev_Y_n);*/
			{
				hipFree(dev_sigmal);
				hipFree(dev_sigmaa);
				hipFree(dev_sigmab);
				hipFree(dev_sigmax);
				hipFree(dev_sigmay);
				hipFree(dev_clustersize);
				//////////////////////////
				hipFree(dev_kseed_sita_n) ;
				hipFree(dev_kseed_m_n);
				hipFree(dev_kseed_L_n);
				hipFree(dev_kseed_X_n);
				hipFree(dev_kseed_Y_n);
			}
		}
}
/*------------------------------------------------------------------------------------------*/
/**
*
*����GPU������ȡ����
*.cu���ص���
*
*@param			 kseedsSize   ���Ӹ���
*@param			 m_width      ͼ����
*@param			 m_height     ͼ��߶�
*@param 	[in]	 host_m_lvec  ����L����
*@param 	[in]	 host_m_avec  ����A����
*@param 	[in]	 host_m_bvec  ����B����
*@param 	[out]	 host_kseedsl  ����L����
*@param 	[out]	 host_kseedsa  ����A����
*@param 	[out]	 host_kseedsb  ����B����
*@param 	[out]	 host_kseedsx  ����X����
*@param 	[out]	 host_kseedsy  ����Y����
*@param 	[in]	 host_klabels  ͼ��ͼ��ı궨ͼ��
*
*/
/*------------------------------------------------------------------------------------------*/
void  GetSeedsThetaMLXY_gpu(
	const int kseedsSize,
	const int m_width,
	const int m_height,
	double*     host_sita_n,
	double*     host_m_n,
	double*     host_L_n,
	double*     host_X_n,
	double*     host_Y_n,
	double*     host_kseedsTheta,
	double*     host_kseedsM,
	double*     host_kseedsL,
	double*     host_kseedsX,
	double*     host_kseedsY,
	int*		host_klabels)
{
	/*		cudaStatus = hipMalloc((void**)&dev_kseed_sita_n, kseedsSize* sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_kseed_m_n, kseedsSize* sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_kseed_L_n, kseedsSize* sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_kseed_X_n, kseedsSize* sizeof(double));
		cudaStatus = hipMalloc((void**)&dev_kseed_Y_n, kseedsSize* sizeof(double));*/

}
/*------------------------------------------------------------------------------------------*/
/**
*����ͼ������Ӿ���\n
*.cu���ص���
*
*@param	[in]	host_labels      labels����
*@param		    NumLabels       label����
*@param	[out]	host_Matrix_E   ���Ӿ���
*@param		    width         ͼ����
*@param		    height        ͼ��߶�
*
*/
/*------------------------------------------------------------------------------------------*/
__global__ void Nighbour_E_matrix_thread(
	int*			labels,
	const  int		NumLabels,
	UINT32 *		Matrix_E,
	const int		width,
	const int		height)
{
#if 0
int k =blockIdx.x*blockDim.x+threadIdx.x;
	int j =blockIdx.y*blockDim.y+threadIdx.y;
	int mainindex = j*width + k;
	const int dx8[8] = {-1, -1,  0,  1, 1, 1, 0, -1};
	const int dy8[8] = { 0, -1, -1, -1, 0, 1, 1,  1};
	////////////////////////////////////////////////
	int cind(0);
//	static int DoCount=0;
	for (register int ii=0;ii<NumLabels;ii++) {
		Matrix_E[ii*NumLabels+ii]=0;
	}

	int np(0);
	for(register  int i = 0; i < 8; i++ ){
		int x = k + dx8[i];
		int y = j + dy8[i];
		if( (x >= 0 && x < width) && (y >= 0 && y <height) ){
			int index = y*width + x;
			if( labels[mainindex] != labels[index] ){
				UINT32  matrix_x=labels[index];
				UINT32  matrix_y=labels[mainindex];
				Matrix_E[matrix_x*NumLabels+matrix_y]=++Matrix_E[matrix_y*NumLabels+matrix_x];
				if (Matrix_E[matrix_x*NumLabels+matrix_y]>999*2){
					Matrix_E[matrix_x*NumLabels+matrix_y]=Matrix_E[matrix_y*NumLabels+matrix_x]=999*2;
				}
			}
		}
	}
	mainindex++;
	for(register  int i = 0; i <NumLabels; i++ ){
		for(register  int j = 0; j <NumLabels; j++ ){			
			Matrix_E[i*NumLabels+j]=Matrix_E[i*NumLabels+j]>0?1:0;
		}
	}
	{
		double  threshold;		//���㳬�������
		threshold=4*sqrt((float)width*height/NumLabels);	//�ܳ�
		threshold=threshold/8; //8��֮һ�ܳ�

		for(register  int i = 0; i <NumLabels; i++ ){
			for(register  int j = 0; j <NumLabels; j++ ){
				if (Matrix_E[i*NumLabels+j]>0){
					Matrix_E[i*NumLabels+j]=1;     
				}else{
					Matrix_E[i*NumLabels+j]=0;    
				}
			}
		}
	}
#endif
	int k =blockIdx.x*blockDim.x+threadIdx.x;
	int j =blockIdx.y*blockDim.y+threadIdx.y;
	int mainindex = j*width + k;
	const int dx8[8] = {-1, -1,  0,  1, 1, 1, 0, -1};
	const int dy8[8] = { 0, -1, -1, -1, 0, 1, 1,  1};
	/*********************************************************************************************/
	if( (k >= 0 && k < width) && (j >= 0 && j <height) ){
			for(register  int i = 0; i < 8; i++ ){
					int x = k + dx8[i];
					int y = j + dy8[i];
					if( (x >= 0 && x < width) && (y >= 0 && y <height) ){
						int index = y*width + x;
						if( labels[mainindex] != labels[index] ){
							UINT32  matrix_x=labels[index];
							UINT32  matrix_y=labels[mainindex];
							Matrix_E[matrix_x*NumLabels+matrix_y]=Matrix_E[matrix_y*NumLabels+matrix_x]=1;
							//if (matrix_x==matrix_y) assert(Matrix_E[matrix_x*NumLabels+matrix_y]==0);
						}
					}
				}
	}
	/*********************************************************************************************/
}
/*------------------------------------------------------------------------------------------*/
/**
*����ͼ������Ӿ���\n
*C��������
*
*@param	[in]	host_labels      labels����
*@param		    NumLabels       label����
*@param	[out]	host_Matrix_E   ���Ӿ���
*@param		    width         ͼ����
*@param		    height        ͼ��߶�
*
*/
/*------------------------------------------------------------------------------------------*/
void Get_Nighbour_E_matrix_gpu(
	int*			host_labels,
	const  int		NumLabels,
	UINT32 *		host_Matrix_E,
	const int		width,
	const int		height)
{
	/*---------------------------*/
	hipError_t cudaStatus;
	int*			dev_labels;
	UINT32 *		dev_Matrix_E;
	/*---------------------------*/
	cudaStatus = hipMalloc((void**)&dev_labels, width*height * sizeof(int));
	cudaStatus = hipMalloc((void**)&dev_Matrix_E, NumLabels*NumLabels* sizeof(UINT32));
	cudaStatus=hipMemcpy(dev_labels,host_labels,width*height*sizeof(int),hipMemcpyHostToDevice);
	cudaStatus=hipMemcpy(dev_Matrix_E,host_Matrix_E,NumLabels*NumLabels*sizeof(UINT32),hipMemcpyHostToDevice);
	/*--------------------------------------------*/
	dim3    threadsPerBlock(16,16);
	dim3    numBlock((width+threadsPerBlock.x-1)/threadsPerBlock.x,(height+threadsPerBlock.y-1)/threadsPerBlock.y);
	 Nighbour_E_matrix_thread<<<numBlock,threadsPerBlock>>>(dev_labels,
		NumLabels,
		dev_Matrix_E,
		width,
		height);
	cudaGetLastError_Sync_CUI();
	/*--------------------------------------------*/
	cudaStatus=hipMemcpy(host_Matrix_E,dev_Matrix_E,NumLabels*NumLabels*sizeof(UINT32),hipMemcpyDeviceToHost);
	hipFree(dev_labels);
	hipFree(dev_Matrix_E);
}
/*------------------------------------------------------------------------------------------*/
/**
*
*
*
*/
/*------------------------------------------------------------------------------------------*/
int GetThreadIdSelfwinlinux(void)
{
		
		  int thread_id=0;
#if _DEBUG
		  int device_count;
		  hipGetDeviceCount(&device_count);
#endif

		
#if _MSC_VER
		  thread_id =::GetCurrentThreadId();
#elif linux||__linux||__linux__||__GNUC__
		  //thread_id=(struct pthread_fake *)pthread_self()->tid;
		  thread_id=syscall( __NR_gettid );
#else
		  thread_id=-1;
		  assert(0);
#endif

	return thread_id;
}
/*------------------------------------------------------------------------------------------*/
/**
*
*
*
*/
/*------------------------------------------------------------------------------------------*/
//int GetCudaNum(){
//
//}

/*------------------------------------------------------------------------------------------*/
/**
*
*
*
*/
/*------------------------------------------------------------------------------------------*/