#include "hip/hip_runtime.h"

#include "hip/device_functions.h"
#include "math_functions.h"

#include <iostream>
#include <fstream>
#include <string>
#include <sstream>

#define  WeightZoom (10000)

#ifndef M_PI
#define M_PI (3.14159265358979323846)
#endif

#include<assert.h>
/*------------------------------------------------------------------------------------------*/
/**
*
*
*
*/
/*------------------------------------------------------------------------------------------*/
#define Unclassify (-2)
#define  Remove (-1)
#if 1
#define  Unknow (0)   
#endif
#define  Ground11  (-11)
#define  Ground12  (-12)
#define  Ground13  (-13)
#define  Ground (1)
#define  GroundCv   cvScalar(Ground,0,0,0)
#define  GroundPending  (-14)
#if 0
#define  NotSky   (1.5)
#endif
#define  Vertical (2)
#define  VerticalCv  cvScalar(Vertical,0,0,0)
#define  VerticalPending (-21)
#if 0
#define  NotGround (2.5)
#endif 
#define  Sky  (3)
#define  SkyCv  cvScalar(Sky,0,0,0)
#define  SkyPending (-31)
#ifdef Vertical 
#if Vertical 
#define Vertical_Tree      (4)
#define Vertical_Building  (5)
#endif
#endif 
/*------------------------------------------------------------------------------------------*/
/**
*
*
*
*/
/*------------------------------------------------------------------------------------------*/
typedef signed int          INT32;
/*------------------------------------------------------------------------------------------*/
/**
*
*
*
*/
/*------------------------------------------------------------------------------------------*/
extern "C" bool cudaInit_CUI(void);
/*------------------------------------------------------------------------------------------*/
/**
*
*
*
*/
/*------------------------------------------------------------------------------------------*/
extern "C" bool cudaDeInit_CUI(void);
/*------------------------------------------------------------------------------------------*/
/**
*
*
*
*/
/*------------------------------------------------------------------------------------------*/
extern "C" void cudaGetLastError_Sync_CUI(void);
/*------------------------------------------------------------------------------------------*/
/**
*
*
*
*/
/*------------------------------------------------------------------------------------------*/
extern "C"  void classify_SkyVerticalGround_gpu(
	double horizontal_line,
	INT32*	 Matrix_Category_Lable_host,
	double* SkyWeightArray_host,
	double* VerticalWeightArray_host,
	double* GroundWeightArray_host,
	double* SkyWeightSp_host,
	double* VerticalWeightSp_host,
	double* GroundWeightSp_host,
	int*    labels_host,
	int     Numlabels,
	int Width,
	int Height);
/*------------------------------------------------------------------------------------------*/
/**
*
*
*
*/
/*------------------------------------------------------------------------------------------*/
//__inline__ __host__ __device__ float GetPosWeightArrayV(float horizontal_line, float n,float sigma,float i)
//{
//	float Weight=(1/(sqrt(2*M_PI)*sigma))*exp(-pow(i,2)/pow(sigma,2))*WeightZoom;
//	return Weight;
//}
/*------------------------------------------------------------------------------------------*/
/**
*
*int i = blockIdx.x * blockDim.x + threadIdx.x;

Read more at: http://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#ixzz3Ts8QtgEN 
Follow us: @GPUComputing on Twitter | NVIDIA on Facebook
*
*/
/*------------------------------------------------------------------------------------------*/
__global__ void	FillWeightArrayV_Kernel(double horizontal_line, double n,double* WeightArray_dev,float Height)
{
	
	int i = blockIdx.x * blockDim.x + threadIdx.x;	
	if (i>Height) return;
	double sigma;
	double L,L_1,L_2;
	L_1=horizontal_line;
	L_2=Height-horizontal_line;
	L=(L_1>L_2)?L_2:L_1;
	//sigma=L/3;
	
	sigma=L/6;//����sigma λ��

	int x=i-L;
	int y=0-x+horizontal_line;	
		
	if (y>=0&&y<Height){
		//WeightArray_dev[y]=(1/(sqrt(2*M_PI)*sigma))*exp(-pow((float)x,2)/pow(sigma,2))*WeightZoom;
		WeightArray_dev[y]=(1/(sqrt(2*M_PI)*sigma))*exp(-pow((float)x,2)/pow(sigma,2))*WeightZoom*2;//2015��5��22��17:31:00 �����ܶȳ��Զ�;
	}
	
}
/*------------------------------------------------------------------------------------------*/
/**
*
*
*
*/
/*------------------------------------------------------------------------------------------*/
__global__ void	FillWeightArrayS_out_Kernel(double horizontal_line, double n,double* WeightArray_dev,float Height)
{
#if 0
int i = blockIdx.x * blockDim.x + threadIdx.x;	
	if (i>Height) return;
	double C_1,C_2;
	double L_1,L_2;
	L_1=Height-horizontal_line;
	L_2=horizontal_line;
	C_1=(2*n+2.0)/((2*n+1)*pow(L_1,(2*n+2)/(2*n+1.0)));
	C_2=(2*n+2.0)/((2*n+1)*pow(L_2,(2*n+2)/(2*n+1.0)));
	/**************************************************/
	int x=i-Height+horizontal_line;
	int y=horizontal_line-x-1;
	/**************************************************/
	if (y>=0&&y<Height){
		if (x<0){
			WeightArray_dev[y]=-1*C_1*pow(-x,1.0/(2*n+1))*WeightZoom;
		}else if (x>=0){
			WeightArray_dev[y]=C_2*pow(x,1.0/(2*n+1))*WeightZoom;
		}else{
			assert(0);
		}
	}
	/**************************************************/
#endif
#if 1
	int i = blockIdx.x * blockDim.x + threadIdx.x;	
	if (i>Height) return;
	double C_1,C_2;
	double L_1,L_2;
	L_1=Height-horizontal_line;
	L_2=horizontal_line;
	C_1=(2*n+2.0)/((2*n+1)*pow(L_1,(2*n+2)/(2*n+1.0)));
	C_2=(2*n+2.0)/((2*n+1)*pow(L_2,(2*n+2)/(2*n+1.0)));
	/**************************************************/
	int x=horizontal_line-i;
	int y=horizontal_line-x-1;
	/**************************************************/
	if (y>=0&&y<Height){
		if (x<0){
			WeightArray_dev[y]=-1*C_1*pow(1.0*-x,1.0/(2*n+1))*WeightZoom;
		}else if (x>=0){
			WeightArray_dev[y]=C_2*pow(1.0*x,1.0/(2*n+1))*WeightZoom;
		}else{
			assert(0);
		}
	}
	/**************************************************/
#endif
}
/*------------------------------------------------------------------------------------------*/
/**
*
*
*
*/
/*------------------------------------------------------------------------------------------*/
__global__ void	FillWeightArrayG_out_Kernel(double horizontal_line, double n,double* WeightArray_dev,float Height)
{

	int i = blockIdx.x * blockDim.x + threadIdx.x;	
	if (i>Height) return;

	double C_1,C_2;
	double L_1,L_2;	
	L_1=Height-horizontal_line;
	L_2=horizontal_line;
	C_1=(2*n+2.0)/((2*n+1)*pow(L_1,(2*n+2)/(2*n+1.0)));
	C_2=(2*n+2.0)/((2*n+1)*pow(L_2,(2*n+2)/(2*n+1.0)));
	
	int x=i-Height+horizontal_line;
	int y=horizontal_line-x-1;

		if (y>=0&&y<Height){
			if (x<0){
				//�溯��
				WeightArray_dev[y]=-1*(-1*C_1*pow(-1.0*x,1.0/(2*n+1)))*WeightZoom;
			}else{
				WeightArray_dev[y]=-1*C_2*pow(1.0*x,1.0/(2*n+1))*WeightZoom;
			}
			/*if (WeightArray_dev[y]<0){
				WeightArray_dev[y]=0;
			}*/
			
		}
}
/*------------------------------------------------------------------------------------------*/
/**
*
*
*
*/
/*------------------------------------------------------------------------------------------*/
 void FillWeightArrayV(double horizontal_line, double n,int Height,double* WeightArray_dev)
{
	
	FillWeightArrayV_Kernel<<<(Height+1023)/1024,1024>>>(horizontal_line,n,WeightArray_dev,Height);
}
/*------------------------------------------------------------------------------------------*/
/**
*
*
*
*/
/*------------------------------------------------------------------------------------------*/
 void FillWeightArrayS_out(double horizontal_line, double n,int Height,double* WeightArray_dev)
 {

	 FillWeightArrayS_out_Kernel<<<(Height+1023)/1024,1024>>>(horizontal_line,n,WeightArray_dev,Height);
 }
/*------------------------------------------------------------------------------------------*/
/**
*
*
*
*/
/*------------------------------------------------------------------------------------------*/
 void FillWeightArrayG_out(double horizontal_line, double n,int Height,double* WeightArray_dev)
 {

	 FillWeightArrayG_out_Kernel<<<(Height+1023)/1024,1024>>>(horizontal_line,n,WeightArray_dev,Height);
 }
 /*------------------------------------------------------------------------------------------*/
 /**
 *
 *
 *
 */
 /*------------------------------------------------------------------------------------------*/
 __global__ void Statistical_SVGsum_Fill_Category_Lable(
	 int    Numlabels,
	 double* SkyWeightSp_dev,
	 double* VerticalWeightSp_dev,
	 double* GroundWeightSp_dev,
	 INT32*	Matrix_Category_Lable_dev,
	 double* SVG_SpSum,
	 double* SkyWeightArray_dev,
	 double* VerticalWeightArray_dev,
	 double* GroundWeightArray_dev,
	 int Height,
	 int Width,
	 int*labels_dev)
 {
 
	 int spi = blockIdx.x * blockDim.x + threadIdx.x;	
	 if (spi<Numlabels){
		 SkyWeightSp_dev[spi]=0;
		 VerticalWeightSp_dev[spi]=0;
		 GroundWeightSp_dev[spi]=0;
		 SVG_SpSum[spi]=0;


	 for (register int x=0;x<Width;x++){
		 for (register int y=0;y<Height;y++){
			
			 int pixelIdx=y*Width+x;

			 if (spi==labels_dev[pixelIdx]){

				  SkyWeightSp_dev[spi]+=SkyWeightArray_dev[y];
				  VerticalWeightSp_dev[spi]+=VerticalWeightArray_dev[y];
				  GroundWeightSp_dev[spi]+=GroundWeightArray_dev[y];
				  SVG_SpSum[spi]+=1;

			 }
			
		 }
	 }
#if 1
	SkyWeightSp_dev[spi]/=SVG_SpSum[spi];
	 VerticalWeightSp_dev[spi]/=SVG_SpSum[spi];
	 GroundWeightSp_dev[spi]/=SVG_SpSum[spi];
	 //////////////////////////////////////////////////////////////
	 if ((SkyWeightSp_dev[spi]>=VerticalWeightSp_dev[spi])
		 &&(SkyWeightSp_dev[spi]>=GroundWeightSp_dev[spi])){
			 Matrix_Category_Lable_dev[spi]=Sky;
	 }
	 //////////////////////////////////////////////////////////////
	 if ((GroundWeightSp_dev[spi]>=VerticalWeightSp_dev[spi])
		 &&(GroundWeightSp_dev[spi]>=SkyWeightSp_dev[spi])){
			 Matrix_Category_Lable_dev[spi]=Ground;
	 }
	 //////////////////////////////////////////////////////////////
	 if ((VerticalWeightSp_dev[spi]>=SkyWeightSp_dev[spi])
		 &&(VerticalWeightSp_dev[spi]>=GroundWeightSp_dev[spi])){
			 Matrix_Category_Lable_dev[spi]=Vertical;
	 }	
	 ////////////////////////////////////////////////////////////
#endif
	 }	
	 
 }
  /*------------------------------------------------------------------------------------------*/
 /**
 *
 *
 *
 */
 /*------------------------------------------------------------------------------------------*/
 void Statistical_SVGsum_Fill_Category_Lable_CPU(
	 int    Numlabels,	
	 INT32*	Matrix_Category_Lable_host,
	 double* SkyWeightArray_host,
	 double* VerticalWeightArray_host,
	 double* GroundWeightArray_host,
	 double* SkyWeightSp_host,
	 double* VerticalWeightSp_host,
	 double* GroundWeightSp_host,
	 int Height,
	 int Width,
	 int*labels_host)
 {
		 hipError_t cudaStatus;
#if _DEBUG
		 cudaStatus = cudaStatus;
#endif
		 double* SVG_SpSum_host=new double[Numlabels];
		 memset(SVG_SpSum_host,0,sizeof(double)*Numlabels);
		 memset(SkyWeightSp_host,0,sizeof(double)*Numlabels);
		 memset(VerticalWeightSp_host,0,sizeof(double)*Numlabels);
		 memset(GroundWeightSp_host,0,sizeof(double)*Numlabels);
		 for (register int x=0;x<Width;x++){
			 for (register int y=0;y<Height;y++){
				 int pixelIdx=y*Width+x;
				 int spi=labels_host[pixelIdx];
				 	 SkyWeightSp_host[spi]+=SkyWeightArray_host[y];
					 VerticalWeightSp_host[spi]+=VerticalWeightArray_host[y];
					 GroundWeightSp_host[spi]+=GroundWeightArray_host[y];
					 SVG_SpSum_host[spi]+=1;
			 }
		 }
		 for (int spi=0;spi<Numlabels;spi++){
			 SkyWeightSp_host[spi]/=SVG_SpSum_host[spi];
			 VerticalWeightSp_host[spi]/=SVG_SpSum_host[spi];
			 GroundWeightSp_host[spi]/=SVG_SpSum_host[spi];
#if 1
			 //////////////////////////////////////////////////////////////
			 if ((SkyWeightSp_host[spi]>=VerticalWeightSp_host[spi])
				 &&(SkyWeightSp_host[spi]>=GroundWeightSp_host[spi])){
					 Matrix_Category_Lable_host[spi]=Sky;
			 }
			 //////////////////////////////////////////////////////////////
			 if ((GroundWeightSp_host[spi]>=VerticalWeightSp_host[spi])
				 &&(GroundWeightSp_host[spi]>=SkyWeightSp_host[spi])){
					 Matrix_Category_Lable_host[spi]=Ground;
			 }
			 //////////////////////////////////////////////////////////////
			 if ((VerticalWeightSp_host[spi]>=SkyWeightSp_host[spi])
				 &&(VerticalWeightSp_host[spi]>=GroundWeightSp_host[spi])){
					 Matrix_Category_Lable_host[spi]=Vertical;
			 }	
			 ////////////////////////////////////////////////////////////
#endif
		 }
		 delete [] SVG_SpSum_host;
		/* delete [] SkyWeightSp_host;
		 delete [] VerticalWeightSp_host;
		 delete [] GroundWeightSp_host;*/
 }
/*------------------------------------------------------------------------------------------*/
/**
*
*
*@Param [out] Matrix_Category_Lable 
*/
/*------------------------------------------------------------------------------------------*/
 __global__ void Fill_Category_Lable(
	 double* SkyWeightSp_dev,
	 double* VerticalWeightSp_dev,
	 double* GroundWeightSp_dev,
	 INT32*	Matrix_Category_Lable)
{
	 int spi = blockIdx.x * blockDim.x + threadIdx.x;	
	  //////////////////////////////////////////////////////////////
	  if ((SkyWeightSp_dev[spi]>=VerticalWeightSp_dev[spi])
		  &&(SkyWeightSp_dev[spi]>=GroundWeightSp_dev[spi])){
			  Matrix_Category_Lable[spi]=Sky;
	  }
	  //////////////////////////////////////////////////////////////
	  if ((GroundWeightSp_dev[spi]>=VerticalWeightSp_dev[spi])
		  &&(GroundWeightSp_dev[spi]>=SkyWeightSp_dev[spi])){
			  Matrix_Category_Lable[spi]=Ground;
	  }
	  //////////////////////////////////////////////////////////////
	  if ((VerticalWeightSp_dev[spi]>=SkyWeightSp_dev[spi])
		  &&(VerticalWeightSp_dev[spi]>=GroundWeightSp_dev[spi])){
			  Matrix_Category_Lable[spi]=Vertical;
	  }	
	  //////////////////////////////////////////////////////////////
}
/*------------------------------------------------------------------------------------------*/
/**
*
*
*
*/
/*------------------------------------------------------------------------------------------*/
 void classify_SkyVerticalGround_gpu(
	 double horizontal_line,
	 INT32*	 Matrix_Category_Lable_host,
	 double* SkyWeightArray_host,
	 double* VerticalWeightArray_host,
	 double* GroundWeightArray_host,
	 double* SkyWeightSp_host,
	 double* VerticalWeightSp_host,
	 double* GroundWeightSp_host,
	 int*    labels_host,
	 int     Numlabels,
	 int Width,
	 int Height)
{
	 hipError_t cudaStatus;
	 double *SkyWeightArray_dev;
	 double *VerticalWeightArray_dev;
	 double *GroundWeightArray_dev;
	 cudaStatus = hipMalloc((void**)&SkyWeightArray_dev,Height * sizeof(double));
	 cudaStatus = hipMalloc((void**)&VerticalWeightArray_dev,Height* sizeof(double));
	 cudaStatus = hipMalloc((void**)&GroundWeightArray_dev,Height * sizeof(double));
	 {
		 /////////////////////////////////////////////////////////////////////////////////
#if 1
 FillWeightArrayG_out(horizontal_line,1,Height,GroundWeightArray_dev);
#endif
 #if 1
 FillWeightArrayV(horizontal_line,1,Height,VerticalWeightArray_dev);
#endif
#if 1
 FillWeightArrayS_out(horizontal_line,1,Height,SkyWeightArray_dev);
#endif
		
		
		
		 /////////////////////////////////////////////////////////////////////////////////
		 double* SkyWeightSp_dev;
		 double* VerticalWeightSp_dev;
		 double* GroundWeightSp_dev;
		 double* SVG_SpSum_dev;
		 int* labels_dev;
		 INT32* Matrix_Category_Lable_dev;
		 //////////////////////////////////////////////////////////////////////////////
		  cudaStatus = hipMalloc((void**)&SkyWeightSp_dev, Numlabels*sizeof(double));
		  cudaStatus = hipMalloc((void**)&VerticalWeightSp_dev,Numlabels*sizeof(double));
		  cudaStatus = hipMalloc((void**)&GroundWeightSp_dev,Numlabels*sizeof(double));
		  cudaStatus = hipMalloc((void**)&SVG_SpSum_dev,Numlabels*sizeof(double));
		  cudaStatus = hipMalloc((void**)&labels_dev,Width*Height * sizeof(int));
		  cudaStatus = hipMalloc((void**)&Matrix_Category_Lable_dev,Numlabels * sizeof(double));
		  hipMemcpy(labels_dev,labels_host,Width*Height*sizeof(int),hipMemcpyHostToDevice);
		  /////////////////////////////////////////////////////////////////////////////
#if 0
Statistical_SVGsum_Fill_Category_Lable<<<(Numlabels+1023)/1024,1024>>>(
								 Numlabels,
								 SkyWeightSp_dev,
								 VerticalWeightSp_dev,
								 GroundWeightSp_dev,
								 Matrix_Category_Lable_dev,
								 SVG_SpSum_dev,
								 SkyWeightArray_dev,
								 VerticalWeightArray_dev,
								 GroundWeightArray_dev,
								 Height,
								 Width,
								 labels_dev);
		 cudaGetLastError_Sync_CUI();
		 cudaMemcpy(Matrix_Category_Lable_host,Matrix_Category_Lable_dev,Numlabels*sizeof(double) ,cudaMemcpyDeviceToHost);
		 cudaGetLastError_Sync_CUI();
#else		 
		 /* double* VerticalWeightArray_host=new double[Height];
		  double* GroundWeightArray_host=new double[Height];
		  double* SkyWeightArray_host=new double[Height];*/
		  cudaStatus=hipMemcpy(VerticalWeightArray_host,VerticalWeightArray_dev,Height*sizeof(double),hipMemcpyDeviceToHost);
		  cudaStatus=hipMemcpy(GroundWeightArray_host,GroundWeightArray_dev,Height*sizeof(double),hipMemcpyDeviceToHost);
		  cudaStatus=hipMemcpy(SkyWeightArray_host,SkyWeightArray_dev,Height*sizeof(double),hipMemcpyDeviceToHost);
#if _MSC_VER&&_DEBUG
 {
			  char data_t[1024];																			
			  std::ofstream outfile;								   
			  outfile.open("Matrix_Weight_GVS_zlm_cuda.data",std::ios::out);
			  for( int i = 0; i <Height; i++ ){
				  {
					   double value_t=i;
					  sprintf_s(data_t,1024," %0.2e ",value_t);
					  outfile<<data_t;		
				  }
				  {
					  double value_t=GroundWeightArray_host[i];
					  sprintf_s(data_t,1024," %0.2e ",value_t);
					  outfile<<data_t;										
				  }
				  {
					  double value_t=VerticalWeightArray_host[i];
					  sprintf_s(data_t,1024," %0.2e ",value_t);
					  outfile<<data_t;										
				  }
				  {
					  double value_t=SkyWeightArray_host[i];
					  sprintf_s(data_t,1024," %0.2e ",value_t);
					  outfile<<data_t;										
				  }
				  outfile<<std::endl;			 
			  } 
			  outfile.close();
		  }
#endif
		 

		  Statistical_SVGsum_Fill_Category_Lable_CPU(
			  Numlabels,	
			  Matrix_Category_Lable_host,
			  SkyWeightArray_host,
			  VerticalWeightArray_host,
			  GroundWeightArray_host,
			  SkyWeightSp_host,
			  VerticalWeightSp_host,
			  GroundWeightSp_host,
			  Height,
			  Width,
			  labels_host);
		  //delete [] SkyWeightArray_host;
		  //delete [] VerticalWeightArray_host;
		  //delete [] GroundWeightArray_host;
#endif		 
		 /////////////////////////////////////////////////
		 hipFree(SkyWeightSp_dev);
		 hipFree(VerticalWeightSp_dev);
		 hipFree(GroundWeightSp_dev);
		 hipFree(SVG_SpSum_dev);
		 hipFree(labels_dev);
		 hipFree(Matrix_Category_Lable_dev);
	 }
	 hipFree(SkyWeightArray_dev);
	 hipFree(VerticalWeightArray_dev);
	 hipFree(GroundWeightArray_dev);
} 
 /*------------------------------------------------------------------------------------------*/
 /**
 *
 *
 *
 */
 /*------------------------------------------------------------------------------------------*/